#include "hip/hip_runtime.h"
/*
 * This file belongs to the Galois project, a C++ library for exploiting parallelism.
 * The code is being released under the terms of the 3-Clause BSD License (a
 * copy is located in LICENSE.txt at the top-level directory).
 *
 * Copyright (C) 2018, The University of Texas at Austin. All rights reserved.
 * UNIVERSITY EXPRESSLY DISCLAIMS ANY AND ALL WARRANTIES CONCERNING THIS
 * SOFTWARE AND DOCUMENTATION, INCLUDING ANY WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR ANY PARTICULAR PURPOSE, NON-INFRINGEMENT AND WARRANTIES OF
 * PERFORMANCE, AND ANY WARRANTY THAT MIGHT OTHERWISE ARISE FROM COURSE OF
 * DEALING OR USAGE OF TRADE.  NO WARRANTY IS EITHER EXPRESS OR IMPLIED WITH
 * RESPECT TO THE USE OF THE SOFTWARE OR DOCUMENTATION. Under no circumstances
 * shall University be liable for incidental, special, indirect, direct or
 * consequential damages or loss of profits, interruption of business, or
 * related expenses which may arise from use of Software or Documentation,
 * including but not limited to those resulting from defects in Software and/or
 * Documentation, or loss or inaccuracy of data of any kind.
 */

/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
#include "gen_cuda.cuh"
static const int __tb_PageRank = TB_SIZE;
static const int __tb_InitializeGraph = TB_SIZE;
__global__ void ResetGraph(CSRGraph graph, unsigned int __begin, unsigned int __end, const float  local_alpha, float * p_delta, uint32_t * p_nout, float * p_residual, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_value[src] = 0;
      p_nout[src] = 0;
      p_delta[src] = 0;
      p_residual[src] = local_alpha;
    }
  }
  // FP: "10 -> 11;
}
__global__ void InitializeGraph(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_nout, DynamicBitset& bitset_nout)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_InitializeGraph;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
    }
    // FP: "9 -> 10;
    // FP: "12 -> 13;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "13 -> 14;
    __shared__ struct { ; } _np_closure [TB_SIZE];
    // FP: "14 -> 15;
    // FP: "15 -> 16;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "18 -> 19;
    // FP: "19 -> 20;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "20 -> 21;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "21 -> 22;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "24 -> 25;
    __syncthreads();
    // FP: "25 -> 26;
    while (true)
    {
      // FP: "26 -> 27;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "29 -> 30;
      __syncthreads();
      // FP: "30 -> 31;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "31 -> 32;
        __syncthreads();
        // FP: "32 -> 33;
        break;
      }
      // FP: "34 -> 35;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "37 -> 38;
      __syncthreads();
      // FP: "38 -> 39;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "39 -> 40;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "42 -> 43;
      assert(nps.tb.src < __kernel_tb_size);
      // FP: "43 -> 44;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicTestAdd(&p_nout[dst], (uint32_t)1);
          bitset_nout.set(dst);
        }
      }
      // FP: "51 -> 52;
      __syncthreads();
    }
    // FP: "53 -> 54;

    // FP: "54 -> 55;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "55 -> 56;
      const int _np_laneid = hipcub::LaneId();
      // FP: "56 -> 57;
      while (__any_sync(0xffffffff, _np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            atomicTestAdd(&p_nout[dst], (uint32_t)1);
            bitset_nout.set(dst);
          }
        }
      }
      // FP: "74 -> 75;
      __syncthreads();
      // FP: "75 -> 76;
    }

    // FP: "76 -> 77;
    __syncthreads();
    // FP: "77 -> 78;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "78 -> 79;
    while (_np.work())
    {
      // FP: "79 -> 80;
      int _np_i =0;
      // FP: "80 -> 81;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "81 -> 82;
      __syncthreads();
      // FP: "82 -> 83;

      // FP: "83 -> 84;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicTestAdd(&p_nout[dst], (uint32_t)1);
          bitset_nout.set(dst);
        }
      }
      // FP: "92 -> 93;
      _np.execute_round_done(ITSIZE);
      // FP: "93 -> 94;
      __syncthreads();
    }
    // FP: "95 -> 96;
    assert(threadIdx.x < __kernel_tb_size);
  }
  // FP: "97 -> 98;
}
__global__ void PageRank_delta(CSRGraph graph, unsigned int __begin, unsigned int __end, const float  local_alpha, float local_tolerance, float * p_delta, uint32_t * p_nout, float * p_residual, float * p_value, HGAccumulator<unsigned int> DGAccumulator_accum)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage DGAccumulator_accum_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  DGAccumulator_accum.thread_entry();
  // FP: "3 -> 4;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_delta[src] = 0;
      if (p_residual[src] > local_tolerance)
      {
        p_value[src] += p_residual[src];
        if (p_nout[src] > 0)
        {
          p_delta[src] = p_residual[src] * (1 - local_alpha) / p_nout[src];
          DGAccumulator_accum.reduce( 1);
        }
        p_residual[src] = 0;
      }
    }
  }
  // FP: "17 -> 18;
  DGAccumulator_accum.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(DGAccumulator_accum_ts);
  // FP: "18 -> 19;
}
__global__ void PageRank(CSRGraph graph, unsigned int __begin, unsigned int __end, float * p_delta, float * p_residual)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_PageRank;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
    }
    // FP: "9 -> 10;
    // FP: "12 -> 13;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "13 -> 14;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "14 -> 15;
    _np_closure[threadIdx.x].src = src;
    // FP: "15 -> 16;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "18 -> 19;
    // FP: "19 -> 20;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "20 -> 21;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "21 -> 22;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "24 -> 25;
    __syncthreads();
    // FP: "25 -> 26;
    while (true)
    {
      // FP: "26 -> 27;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "29 -> 30;
      __syncthreads();
      // FP: "30 -> 31;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "31 -> 32;
        __syncthreads();
        // FP: "32 -> 33;
        break;
      }
      // FP: "34 -> 35;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "37 -> 38;
      __syncthreads();
      // FP: "38 -> 39;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "39 -> 40;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "42 -> 43;
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "43 -> 44;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          if (p_delta[dst] > 0)
          {
            atomicTestAdd(&p_residual[src], p_delta[dst]);
          }
        }
      }
      // FP: "53 -> 54;
      __syncthreads();
    }
    // FP: "55 -> 56;

    // FP: "56 -> 57;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "57 -> 58;
      const int _np_laneid = hipcub::LaneId();
      // FP: "58 -> 59;
      while (__any_sync(0xffffffff, _np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            if (p_delta[dst] > 0)
            {
              atomicTestAdd(&p_residual[src], p_delta[dst]);
            }
          }
        }
      }
      // FP: "78 -> 79;
      __syncthreads();
      // FP: "79 -> 80;
    }

    // FP: "80 -> 81;
    __syncthreads();
    // FP: "81 -> 82;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "82 -> 83;
    while (_np.work())
    {
      // FP: "83 -> 84;
      int _np_i =0;
      // FP: "84 -> 85;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "85 -> 86;
      __syncthreads();
      // FP: "86 -> 87;

      // FP: "87 -> 88;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          if (p_delta[dst] > 0)
          {
            atomicTestAdd(&p_residual[src], p_delta[dst]);
          }
        }
      }
      // FP: "98 -> 99;
      _np.execute_round_done(ITSIZE);
      // FP: "99 -> 100;
      __syncthreads();
    }
    // FP: "101 -> 102;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "103 -> 104;
}
__global__ void PageRankSanity(CSRGraph graph, unsigned int __begin, unsigned int __end, float local_tolerance, float * p_residual, float * p_value, HGAccumulator<uint64_t> DGAccumulator_residual_over_tolerance, HGAccumulator<float> DGAccumulator_sum, HGAccumulator<float> DGAccumulator_sum_residual, HGReduceMax<float> max_residual, HGReduceMax<float> max_value, HGReduceMin<float> min_residual, HGReduceMin<float> min_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<uint64_t, TB_SIZE>::TempStorage DGAccumulator_residual_over_tolerance_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage DGAccumulator_sum_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage DGAccumulator_sum_residual_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage max_residual_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage max_value_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage min_residual_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage min_value_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  DGAccumulator_residual_over_tolerance.thread_entry();
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  DGAccumulator_sum.thread_entry();
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  DGAccumulator_sum_residual.thread_entry();
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  max_residual.thread_entry();
  // FP: "9 -> 10;
  // FP: "10 -> 11;
  max_value.thread_entry();
  // FP: "11 -> 12;
  // FP: "12 -> 13;
  min_residual.thread_entry();
  // FP: "13 -> 14;
  // FP: "14 -> 15;
  min_value.thread_entry();
  // FP: "15 -> 16;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      max_value.reduce(p_value[src]);
      min_value.reduce(p_value[src]);
      max_residual.reduce(p_residual[src]);
      min_residual.reduce(p_residual[src]);
      DGAccumulator_sum.reduce( p_value[src]);
      DGAccumulator_sum.reduce( p_residual[src]);
      if (p_residual[src] > local_tolerance)
      {
        DGAccumulator_residual_over_tolerance.reduce( 1);
      }
    }
  }
  // FP: "29 -> 30;
  DGAccumulator_residual_over_tolerance.thread_exit<hipcub::BlockReduce<uint64_t, TB_SIZE> >(DGAccumulator_residual_over_tolerance_ts);
  // FP: "30 -> 31;
  DGAccumulator_sum.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(DGAccumulator_sum_ts);
  // FP: "31 -> 32;
  DGAccumulator_sum_residual.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(DGAccumulator_sum_residual_ts);
  // FP: "32 -> 33;
  max_residual.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(max_residual_ts);
  // FP: "33 -> 34;
  max_value.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(max_value_ts);
  // FP: "34 -> 35;
  min_residual.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(min_residual_ts);
  // FP: "35 -> 36;
  min_value.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(min_value_ts);
  // FP: "36 -> 37;
}
void ResetGraph_cuda(unsigned int  __begin, unsigned int  __end, const float & local_alpha, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  ResetGraph <<<blocks, threads>>>(ctx->gg, __begin, __end, local_alpha, ctx->delta.data.gpu_wr_ptr(), ctx->nout.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void ResetGraph_allNodes_cuda(const float & local_alpha, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->gg.nnodes, local_alpha, ctx);
  // FP: "2 -> 3;
}
void ResetGraph_masterNodes_cuda(const float & local_alpha, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, local_alpha, ctx);
  // FP: "2 -> 3;
}
void ResetGraph_nodesWithEdges_cuda(const float & local_alpha, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->numNodesWithEdges, local_alpha, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, __tb_InitializeGraph>>>(ctx->gg, __begin, __end, ctx->nout.data.gpu_wr_ptr(), *(ctx->nout.is_updated.gpu_rd_ptr()));
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void PageRank_delta_cuda(unsigned int  __begin, unsigned int  __end, unsigned int & DGAccumulator_accum, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<unsigned int> _DGAccumulator_accum;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<unsigned int> DGAccumulator_accumval  = Shared<unsigned int>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(DGAccumulator_accumval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _DGAccumulator_accum.rv = DGAccumulator_accumval.gpu_wr_ptr();
  // FP: "8 -> 9;
  PageRank_delta <<<blocks, threads>>>(ctx->gg, __begin, __end, local_alpha, local_tolerance, ctx->delta.data.gpu_wr_ptr(), ctx->nout.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr(), _DGAccumulator_accum);
  // FP: "9 -> 10;
  check_cuda_kernel;
  // FP: "10 -> 11;
  DGAccumulator_accum = *(DGAccumulator_accumval.cpu_rd_ptr());
  // FP: "11 -> 12;
}
void PageRank_delta_allNodes_cuda(unsigned int & DGAccumulator_accum, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_delta_cuda(0, ctx->gg.nnodes, DGAccumulator_accum, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRank_delta_masterNodes_cuda(unsigned int & DGAccumulator_accum, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_delta_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, DGAccumulator_accum, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRank_delta_nodesWithEdges_cuda(unsigned int & DGAccumulator_accum, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_delta_cuda(0, ctx->numNodesWithEdges, DGAccumulator_accum, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRank_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  PageRank <<<blocks, __tb_PageRank>>>(ctx->gg, __begin, __end, ctx->delta.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void PageRank_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void PageRank_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void PageRank_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void PageRankSanity_cuda(unsigned int  __begin, unsigned int  __end, uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<uint64_t> _DGAccumulator_residual_over_tolerance;
  HGAccumulator<float> _DGAccumulator_sum;
  HGAccumulator<float> _DGAccumulator_sum_residual;
  HGReduceMax<float> _max_residual;
  HGReduceMax<float> _max_value;
  HGReduceMin<float> _min_residual;
  HGReduceMin<float> _min_value;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<uint64_t> DGAccumulator_residual_over_toleranceval  = Shared<uint64_t>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(DGAccumulator_residual_over_toleranceval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _DGAccumulator_residual_over_tolerance.rv = DGAccumulator_residual_over_toleranceval.gpu_wr_ptr();
  // FP: "8 -> 9;
  Shared<float> DGAccumulator_sumval  = Shared<float>(1);
  // FP: "9 -> 10;
  // FP: "10 -> 11;
  *(DGAccumulator_sumval.cpu_wr_ptr()) = 0;
  // FP: "11 -> 12;
  _DGAccumulator_sum.rv = DGAccumulator_sumval.gpu_wr_ptr();
  // FP: "12 -> 13;
  Shared<float> DGAccumulator_sum_residualval  = Shared<float>(1);
  // FP: "13 -> 14;
  // FP: "14 -> 15;
  *(DGAccumulator_sum_residualval.cpu_wr_ptr()) = 0;
  // FP: "15 -> 16;
  _DGAccumulator_sum_residual.rv = DGAccumulator_sum_residualval.gpu_wr_ptr();
  // FP: "16 -> 17;
  Shared<float> max_residualval  = Shared<float>(1);
  // FP: "17 -> 18;
  // FP: "18 -> 19;
  *(max_residualval.cpu_wr_ptr()) = 0;
  // FP: "19 -> 20;
  _max_residual.rv = max_residualval.gpu_wr_ptr();
  // FP: "20 -> 21;
  Shared<float> max_valueval  = Shared<float>(1);
  // FP: "21 -> 22;
  // FP: "22 -> 23;
  *(max_valueval.cpu_wr_ptr()) = 0;
  // FP: "23 -> 24;
  _max_value.rv = max_valueval.gpu_wr_ptr();
  // FP: "24 -> 25;
  Shared<float> min_residualval  = Shared<float>(1);
  // FP: "25 -> 26;
  // FP: "26 -> 27;
  *(min_residualval.cpu_wr_ptr()) = 0;
  // FP: "27 -> 28;
  _min_residual.rv = min_residualval.gpu_wr_ptr();
  // FP: "28 -> 29;
  Shared<float> min_valueval  = Shared<float>(1);
  // FP: "29 -> 30;
  // FP: "30 -> 31;
  *(min_valueval.cpu_wr_ptr()) = 0;
  // FP: "31 -> 32;
  _min_value.rv = min_valueval.gpu_wr_ptr();
  // FP: "32 -> 33;
  PageRankSanity <<<blocks, threads>>>(ctx->gg, __begin, __end, local_tolerance, ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr(), _DGAccumulator_residual_over_tolerance, _DGAccumulator_sum, _DGAccumulator_sum_residual, _max_residual, _max_value, _min_residual, _min_value);
  // FP: "33 -> 34;
  check_cuda_kernel;
  // FP: "34 -> 35;
  DGAccumulator_residual_over_tolerance = *(DGAccumulator_residual_over_toleranceval.cpu_rd_ptr());
  // FP: "35 -> 36;
  DGAccumulator_sum = *(DGAccumulator_sumval.cpu_rd_ptr());
  // FP: "36 -> 37;
  DGAccumulator_sum_residual = *(DGAccumulator_sum_residualval.cpu_rd_ptr());
  // FP: "37 -> 38;
  max_residual = *(max_residualval.cpu_rd_ptr());
  // FP: "38 -> 39;
  max_value = *(max_valueval.cpu_rd_ptr());
  // FP: "39 -> 40;
  min_residual = *(min_residualval.cpu_rd_ptr());
  // FP: "40 -> 41;
  min_value = *(min_valueval.cpu_rd_ptr());
  // FP: "41 -> 42;
}
void PageRankSanity_allNodes_cuda(uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRankSanity_cuda(0, ctx->gg.nnodes, DGAccumulator_residual_over_tolerance, DGAccumulator_sum, DGAccumulator_sum_residual, max_residual, max_value, min_residual, min_value, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRankSanity_masterNodes_cuda(uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRankSanity_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, DGAccumulator_residual_over_tolerance, DGAccumulator_sum, DGAccumulator_sum_residual, max_residual, max_value, min_residual, min_value, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRankSanity_nodesWithEdges_cuda(uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRankSanity_cuda(0, ctx->numNodesWithEdges, DGAccumulator_residual_over_tolerance, DGAccumulator_sum, DGAccumulator_sum_residual, max_residual, max_value, min_residual, min_value, local_tolerance, ctx);
  // FP: "2 -> 3;
}
