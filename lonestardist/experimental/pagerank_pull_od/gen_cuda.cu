#include "hip/hip_runtime.h"
/*
 * This file belongs to the Galois project, a C++ library for exploiting parallelism.
 * The code is being released under the terms of the 3-Clause BSD License (a
 * copy is located in LICENSE.txt at the top-level directory).
 *
 * Copyright (C) 2018, The University of Texas at Austin. All rights reserved.
 * UNIVERSITY EXPRESSLY DISCLAIMS ANY AND ALL WARRANTIES CONCERNING THIS
 * SOFTWARE AND DOCUMENTATION, INCLUDING ANY WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR ANY PARTICULAR PURPOSE, NON-INFRINGEMENT AND WARRANTIES OF
 * PERFORMANCE, AND ANY WARRANTY THAT MIGHT OTHERWISE ARISE FROM COURSE OF
 * DEALING OR USAGE OF TRADE.  NO WARRANTY IS EITHER EXPRESS OR IMPLIED WITH
 * RESPECT TO THE USE OF THE SOFTWARE OR DOCUMENTATION. Under no circumstances
 * shall University be liable for incidental, special, indirect, direct or
 * consequential damages or loss of profits, interruption of business, or
 * related expenses which may arise from use of Software or Documentation,
 * including but not limited to those resulting from defects in Software and/or
 * Documentation, or loss or inaccuracy of data of any kind.
 */

/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
float * P_DELTA;
uint32_t * P_NOUT;
float * P_RESIDUAL;
float * P_VALUE;
#include "gen_cuda.cuh"
static const int __tb_PageRank = TB_SIZE;
static const int __tb_InitializeGraph = TB_SIZE;
__global__ void ResetGraph(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float local_alpha, float * p_delta, uint32_t * p_nout, float * p_residual, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_value[src] = 0;
      p_delta[src] = 0;
      p_residual[src] = local_alpha;
      p_nout[src] = 0;
    }
  }
  // FP: "10 -> 11;
}
__global__ void InitializeGraph(CSRGraph graph, DynamicBitset *nout_is_updated, unsigned int __nowned, unsigned int __begin, unsigned int __end, float * p_delta, uint32_t * p_nout, float * p_residual, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_InitializeGraph;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    // FP: "12 -> 13;
    // FP: "15 -> 16;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "16 -> 17;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "19 -> 20;
    // FP: "20 -> 21;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "21 -> 22;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "22 -> 23;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "25 -> 26;
    __syncthreads();
    // FP: "26 -> 27;
    while (true)
    {
      // FP: "27 -> 28;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "30 -> 31;
      __syncthreads();
      // FP: "31 -> 32;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "32 -> 33;
        __syncthreads();
        // FP: "33 -> 34;
        break;
      }
      // FP: "35 -> 36;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "38 -> 39;
      __syncthreads();
      // FP: "39 -> 40;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "40 -> 41;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "43 -> 44;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicAdd(&p_nout[dst], 1);
          nout_is_updated->set(dst);
        }
      }
      // FP: "50 -> 51;
      __syncthreads();
    }
    // FP: "52 -> 53;

    // FP: "53 -> 54;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "54 -> 55;
      const int _np_laneid = hipcub::LaneId();
      // FP: "55 -> 56;
      while (__any_sync(0xffffffff, _np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;

          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            atomicAdd(&p_nout[dst], 1);
            nout_is_updated->set(dst);
          }
        }
      }
      // FP: "71 -> 72;
      __syncthreads();
      // FP: "72 -> 73;
    }

    // FP: "73 -> 74;
    __syncthreads();
    // FP: "74 -> 75;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "75 -> 76;
    while (_np.work())
    {
      // FP: "76 -> 77;
      int _np_i =0;
      // FP: "77 -> 78;
      _np.inspect(nps.fg.itvalue, ITSIZE);
      // FP: "78 -> 79;
      __syncthreads();
      // FP: "79 -> 80;

      // FP: "80 -> 81;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicAdd(&p_nout[dst], 1);
          nout_is_updated->set(dst);
        }
      }
      // FP: "87 -> 88;
      _np.execute_round_done(ITSIZE);
      // FP: "88 -> 89;
      __syncthreads();
    }
  }
  // FP: "91 -> 92;
}
__global__ void PageRank_delta(CSRGraph graph, unsigned int __nowned, unsigned int __begin, unsigned int __end, const float  local_alpha, float local_tolerance, float * p_delta, uint32_t * p_nout, float * p_residual, float * p_value, HGAccumulator<int> ret_val)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  typedef hipcub::BlockReduce<int, TB_SIZE> _br;
  __shared__ _br::TempStorage _ts;
  ret_val.thread_entry();
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_delta[src] = 0;
      if (p_residual[src] > local_tolerance)
      {
        p_value[src] += p_residual[src];
        if (p_nout[src] > 0)
        {
          p_delta[src] = p_residual[src]*(1-local_alpha)/p_nout[src];
          ret_val.reduce( 1);
        }
        p_residual[src] = 0;
      }
    }
  }
  ret_val.thread_exit<_br>(_ts);
}
// TODO: cpu version accumulates sum into local variable then adds all at once
__global__ void PageRank(CSRGraph graph, DynamicBitset *is_updated, unsigned int __nowned, unsigned int __begin, unsigned int __end, float * p_delta, float * p_residual)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_PageRank;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end;
    // FP: "7 -> 8;
    if (pop)
    {
    }
    // FP: "9 -> 10;
    // FP: "12 -> 13;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "13 -> 14;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "14 -> 15;
    _np_closure[threadIdx.x].src = src;
    // FP: "15 -> 16;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "18 -> 19;
    // FP: "19 -> 20;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "20 -> 21;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "21 -> 22;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "24 -> 25;
    __syncthreads();
    // FP: "25 -> 26;
    while (true)
    {
      // FP: "26 -> 27;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "29 -> 30;
      __syncthreads();
      // FP: "30 -> 31;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "31 -> 32;
        __syncthreads();
        // FP: "32 -> 33;
        break;
      }
      // FP: "34 -> 35;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "37 -> 38;
      __syncthreads();
      // FP: "38 -> 39;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "39 -> 40;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "42 -> 43;
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "43 -> 44;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          if (p_delta[dst] > 0)
          {
            atomicAdd(&p_residual[src], p_delta[dst]);
            is_updated->set(src);
          }
        }
      }
      // FP: "52 -> 53;
      __syncthreads();
    }
    // FP: "54 -> 55;

    // FP: "55 -> 56;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "56 -> 57;
      const int _np_laneid = hipcub::LaneId();
      // FP: "57 -> 58;
      while (__any_sync(0xffffffff, _np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            if (p_delta[dst] > 0)
            {
              atomicAdd(&p_residual[src], p_delta[dst]);
              is_updated->set(src);
            }
          }
        }
      }
      // FP: "76 -> 77;
      __syncthreads();
      // FP: "77 -> 78;
    }

    // FP: "78 -> 79;
    __syncthreads();
    // FP: "79 -> 80;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "80 -> 81;
    while (_np.work())
    {
      // FP: "81 -> 82;
      int _np_i =0;
      // FP: "82 -> 83;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "83 -> 84;
      __syncthreads();
      // FP: "84 -> 85;

      // FP: "85 -> 86;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          if (p_delta[dst] > 0)
          {
            atomicAdd(&p_residual[src], p_delta[dst]);
            is_updated->set(src);
          }
        }
      }
      // FP: "95 -> 96;
      _np.execute_round_done(ITSIZE);
      // FP: "96 -> 97;
      __syncthreads();
    }
    // FP: "98 -> 99;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "100 -> 101;
}
void ResetGraph_cuda(unsigned int  __begin, unsigned int  __end, const float & local_alpha, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  ResetGraph <<<blocks, threads>>>(ctx->gg, ctx->numNodesWithEdges, __begin, __end, local_alpha, ctx->delta.data.gpu_wr_ptr(), ctx->nout.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void ResetGraph_all_cuda(const float & local_alpha, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->numNodesWithEdges, local_alpha, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, __tb_InitializeGraph>>>(ctx->gg, ctx->nout.is_updated.gpu_rd_ptr(), ctx->numNodesWithEdges, __begin, __end, ctx->delta.data.gpu_wr_ptr(), ctx->nout.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_all_cuda(struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void PageRank_delta_cuda(unsigned int  __begin, unsigned int  __end, int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<int> retval = Shared<int>(1);
  HGAccumulator<int> _rv;
  *(retval.cpu_wr_ptr()) = 0;
  _rv.rv = retval.gpu_wr_ptr();
  PageRank_delta <<<blocks, threads>>>(ctx->gg, ctx->numNodesWithEdges, __begin, __end, local_alpha, local_tolerance, ctx->delta.data.gpu_wr_ptr(), ctx->nout.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr(), _rv);
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
  __retval = *(retval.cpu_rd_ptr());
  // FP: "7 -> 8;
}
void PageRank_delta_all_cuda(int & __retval, const float & local_alpha, float local_tolerance, struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  PageRank_delta_cuda(0, ctx->numNodesWithEdges, __retval, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRank_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context * ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  PageRank <<<blocks, __tb_PageRank>>>(ctx->gg, ctx->residual.is_updated.gpu_rd_ptr(), ctx->numNodesWithEdges, __begin, __end, ctx->delta.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr());
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void PageRank_all_cuda(struct CUDA_Context * ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
