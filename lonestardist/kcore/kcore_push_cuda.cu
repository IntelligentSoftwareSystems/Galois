#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"
#include "hipcub/hipcub.hpp"
#include "cub/util_allocator.cuh"
#include "thread_work.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ tb_lb=False $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=False $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ dyn_lb=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
struct ThreadWork t_work;
bool enable_lb = true;
#include "kcore_push_cuda.cuh"
static const int __tb_InitializeGraph2 = TB_SIZE;
static const int __tb_KCoreStep1 = TB_SIZE;
__global__ void InitializeGraph2_TB_LB(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_current_degree, DynamicBitset& bitset_current_degree, int * thread_prefix_work_wl, unsigned int num_items, PipeContextT<Worklist2> thread_src_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ unsigned int total_work;
  __shared__ unsigned block_start_src_index;
  __shared__ unsigned block_end_src_index;
  unsigned my_work;
  unsigned src;
  unsigned int offset;
  unsigned int current_work;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  unsigned blockdim_x = BLOCK_DIM_X;
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  // FP: "9 -> 10;
  total_work = thread_prefix_work_wl[num_items - 1];
  // FP: "10 -> 11;
  my_work = ceilf((float)(total_work) / (float) nthreads);
  // FP: "11 -> 12;

  // FP: "12 -> 13;
  __syncthreads();
  // FP: "13 -> 14;

  // FP: "14 -> 15;
  if (my_work != 0)
  {
    current_work = tid;
  }
  // FP: "17 -> 18;
  for (unsigned i =0; i < my_work; i++)
  {
    unsigned int block_start_work;
    unsigned int block_end_work;
    if (threadIdx.x == 0)
    {
      if (current_work < total_work)
      {
        block_start_work = current_work;
        block_end_work=current_work + blockdim_x - 1;
        if (block_end_work >= total_work)
        {
          block_end_work = total_work - 1;
        }
        block_start_src_index = compute_src_and_offset(0, num_items - 1,  block_start_work+1, thread_prefix_work_wl, num_items,offset);
        block_end_src_index = compute_src_and_offset(0, num_items - 1, block_end_work+1, thread_prefix_work_wl, num_items, offset);
      }
    }
    __syncthreads();

    if (current_work < total_work)
    {
      unsigned src_index;
      index_type current_edge;
      src_index = compute_src_and_offset(block_start_src_index, block_end_src_index, current_work+1, thread_prefix_work_wl,num_items, offset);
      src= thread_src_wl.in_wl().dwl[src_index];
      current_edge = (graph).getFirstEdge(src)+ offset;
      {
        index_type dest_node;
        dest_node = graph.getAbsDestination(current_edge);
        atomicTestAdd(&p_current_degree[dest_node], (uint32_t)1);
        bitset_current_degree.set(dest_node);
      }
      current_work = current_work + nthreads;
    }
    __syncthreads();
  }
  // FP: "44 -> 45;
}
__global__ void InitializeGraph2(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_current_degree, DynamicBitset& bitset_current_degree, PipeContextT<Worklist2> thread_work_wl, PipeContextT<Worklist2> thread_src_wl, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_InitializeGraph2;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  unsigned d_limit = DEGREE_LIMIT;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    int index;
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end && ((( src < (graph).nnodes )) ? true: false);
    // FP: "7 -> 8;
    if (pop)
    {
    }
    // FP: "9 -> 10;
    // FP: "12 -> 13;
    // FP: "13 -> 14;
    int threshold = TOTAL_THREADS_1D;
    // FP: "14 -> 15;
    if (pop && (graph).getOutDegree(src) >= threshold)
    {
      index = thread_work_wl.in_wl().push_range(1) ;
      thread_src_wl.in_wl().push_range(1);
      thread_work_wl.in_wl().dwl[index] = (graph).getOutDegree(src);
      thread_src_wl.in_wl().dwl[index] = src;
      pop = false;
    }
    // FP: "17 -> 18;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "18 -> 19;
    __shared__ struct { ; } _np_closure [TB_SIZE];
    // FP: "19 -> 20;
    // FP: "20 -> 21;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "23 -> 24;
    // FP: "24 -> 25;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "25 -> 26;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "26 -> 27;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "29 -> 30;
    __syncthreads();
    // FP: "30 -> 31;
    while (true)
    {
      // FP: "31 -> 32;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "34 -> 35;
      __syncthreads();
      // FP: "35 -> 36;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "36 -> 37;
        __syncthreads();
        // FP: "37 -> 38;
        break;
      }
      // FP: "39 -> 40;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "42 -> 43;
      __syncthreads();
      // FP: "43 -> 44;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "44 -> 45;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "47 -> 48;
      assert(nps.tb.src < __kernel_tb_size);
      // FP: "48 -> 49;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type current_edge;
        current_edge = ns +_np_j;
        {
          index_type dest_node;
          dest_node = graph.getAbsDestination(current_edge);
          atomicTestAdd(&p_current_degree[dest_node], (uint32_t)1);
          bitset_current_degree.set(dest_node);
        }
      }
      // FP: "56 -> 57;
      __syncthreads();
    }
    // FP: "58 -> 59;

    // FP: "59 -> 60;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "60 -> 61;
      const int _np_laneid = hipcub::LaneId();
      // FP: "61 -> 62;
      while (__any_sync(0xffffffff, _np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type current_edge;
          current_edge = _np_w_start +_np_ii;
          {
            index_type dest_node;
            dest_node = graph.getAbsDestination(current_edge);
            atomicTestAdd(&p_current_degree[dest_node], (uint32_t)1);
            bitset_current_degree.set(dest_node);
          }
        }
      }
      // FP: "79 -> 80;
      __syncthreads();
      // FP: "80 -> 81;
    }

    // FP: "81 -> 82;
    __syncthreads();
    // FP: "82 -> 83;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "83 -> 84;
    while (_np.work())
    {
      // FP: "84 -> 85;
      int _np_i =0;
      // FP: "85 -> 86;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "86 -> 87;
      __syncthreads();
      // FP: "87 -> 88;

      // FP: "88 -> 89;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type current_edge;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        current_edge= nps.fg.itvalue[_np_i];
        {
          index_type dest_node;
          dest_node = graph.getAbsDestination(current_edge);
          atomicTestAdd(&p_current_degree[dest_node], (uint32_t)1);
          bitset_current_degree.set(dest_node);
        }
      }
      // FP: "97 -> 98;
      _np.execute_round_done(ITSIZE);
      // FP: "98 -> 99;
      __syncthreads();
    }
    // FP: "100 -> 101;
    assert(threadIdx.x < __kernel_tb_size);
  }
  // FP: "102 -> 103;
}
__global__ void InitializeGraph1(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_current_degree, uint8_t * p_flag, uint32_t * p_trim)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_flag[src]           = true;
      p_trim[src]           = 0;
      p_current_degree[src] = 0;
    }
  }
  // FP: "9 -> 10;
}
__global__ void KCoreStep2(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_current_degree, uint8_t * p_flag, uint32_t * p_trim)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      if (p_flag[src])
      {
        if (p_trim[src] > 0)
        {
          p_current_degree[src] = p_current_degree[src] - p_trim[src];
        }
      }
      p_trim[src] = 0;
    }
  }
  // FP: "12 -> 13;
}
__global__ void KCoreStep1_TB_LB(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t local_k_core_num, uint32_t * p_current_degree, uint8_t * p_flag, uint32_t * p_trim, DynamicBitset& bitset_trim, HGAccumulator<unsigned int> active_vertices, int * thread_prefix_work_wl, unsigned int num_items, PipeContextT<Worklist2> thread_src_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ unsigned int total_work;
  __shared__ unsigned block_start_src_index;
  __shared__ unsigned block_end_src_index;
  unsigned my_work;
  unsigned src;
  unsigned int offset;
  unsigned int current_work;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  unsigned blockdim_x = BLOCK_DIM_X;
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  // FP: "9 -> 10;
  total_work = thread_prefix_work_wl[num_items - 1];
  // FP: "10 -> 11;
  my_work = ceilf((float)(total_work) / (float) nthreads);
  // FP: "11 -> 12;

  // FP: "12 -> 13;
  __syncthreads();
  // FP: "13 -> 14;

  // FP: "14 -> 15;
  if (my_work != 0)
  {
    current_work = tid;
  }
  // FP: "17 -> 18;
  for (unsigned i =0; i < my_work; i++)
  {
    unsigned int block_start_work;
    unsigned int block_end_work;
    if (threadIdx.x == 0)
    {
      if (current_work < total_work)
      {
        block_start_work = current_work;
        block_end_work=current_work + blockdim_x - 1;
        if (block_end_work >= total_work)
        {
          block_end_work = total_work - 1;
        }
        block_start_src_index = compute_src_and_offset(0, num_items - 1,  block_start_work+1, thread_prefix_work_wl, num_items,offset);
        block_end_src_index = compute_src_and_offset(0, num_items - 1, block_end_work+1, thread_prefix_work_wl, num_items, offset);
      }
    }
    __syncthreads();

    if (current_work < total_work)
    {
      unsigned src_index;
      index_type current_edge;
      src_index = compute_src_and_offset(block_start_src_index, block_end_src_index, current_work+1, thread_prefix_work_wl,num_items, offset);
      src= thread_src_wl.in_wl().dwl[src_index];
      current_edge = (graph).getFirstEdge(src)+ offset;
      {
        index_type dst;
        dst = graph.getAbsDestination(current_edge);
        atomicTestAdd(&p_trim[dst], (uint32_t)1);
        bitset_trim.set(dst);
      }
      current_work = current_work + nthreads;
    }
    __syncthreads();
  }
  // FP: "44 -> 45;
}
__global__ void KCoreStep1(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t local_k_core_num, uint32_t * p_current_degree, uint8_t * p_flag, uint32_t * p_trim, DynamicBitset& bitset_trim, HGAccumulator<unsigned int> active_vertices, PipeContextT<Worklist2> thread_work_wl, PipeContextT<Worklist2> thread_src_wl, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_KCoreStep1;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage active_vertices_ts;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  unsigned d_limit = DEGREE_LIMIT;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  active_vertices.thread_entry();
  // FP: "7 -> 8;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    int index;
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "8 -> 9;
    bool pop  = src < __end && ((( src < (graph).nnodes )) ? true: false);
    // FP: "9 -> 10;
    if (pop)
    {
      if (p_flag[src])
      {
        if (p_current_degree[src] < local_k_core_num)
        {
          p_flag[src] = false;
          active_vertices.reduce( 1);
        }
        else
        {
          pop = false;
        }
      }
      else
      {
        pop = false;
      }
    }
    // FP: "17 -> 18;
    // FP: "20 -> 21;
    // FP: "21 -> 22;
    int threshold = TOTAL_THREADS_1D;
    // FP: "22 -> 23;
    if (pop && (graph).getOutDegree(src) >= threshold)
    {
      index = thread_work_wl.in_wl().push_range(1) ;
      thread_src_wl.in_wl().push_range(1);
      thread_work_wl.in_wl().dwl[index] = (graph).getOutDegree(src);
      thread_src_wl.in_wl().dwl[index] = src;
      pop = false;
    }
    // FP: "25 -> 26;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "26 -> 27;
    __shared__ struct { ; } _np_closure [TB_SIZE];
    // FP: "27 -> 28;
    // FP: "28 -> 29;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "31 -> 32;
    // FP: "32 -> 33;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "33 -> 34;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "34 -> 35;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "37 -> 38;
    __syncthreads();
    // FP: "38 -> 39;
    while (true)
    {
      // FP: "39 -> 40;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "42 -> 43;
      __syncthreads();
      // FP: "43 -> 44;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "44 -> 45;
        __syncthreads();
        // FP: "45 -> 46;
        break;
      }
      // FP: "47 -> 48;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "50 -> 51;
      __syncthreads();
      // FP: "51 -> 52;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "52 -> 53;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "55 -> 56;
      assert(nps.tb.src < __kernel_tb_size);
      // FP: "56 -> 57;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type current_edge;
        current_edge = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(current_edge);
          atomicTestAdd(&p_trim[dst], (uint32_t)1);
          bitset_trim.set(dst);
        }
      }
      // FP: "64 -> 65;
      __syncthreads();
    }
    // FP: "66 -> 67;

    // FP: "67 -> 68;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "68 -> 69;
      const int _np_laneid = hipcub::LaneId();
      // FP: "69 -> 70;
      while (__any_sync(0xffffffff, _np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type current_edge;
          current_edge = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(current_edge);
            atomicTestAdd(&p_trim[dst], (uint32_t)1);
            bitset_trim.set(dst);
          }
        }
      }
      // FP: "87 -> 88;
      __syncthreads();
      // FP: "88 -> 89;
    }

    // FP: "89 -> 90;
    __syncthreads();
    // FP: "90 -> 91;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "91 -> 92;
    while (_np.work())
    {
      // FP: "92 -> 93;
      int _np_i =0;
      // FP: "93 -> 94;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "94 -> 95;
      __syncthreads();
      // FP: "95 -> 96;

      // FP: "96 -> 97;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type current_edge;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        current_edge= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(current_edge);
          atomicTestAdd(&p_trim[dst], (uint32_t)1);
          bitset_trim.set(dst);
        }
      }
      // FP: "105 -> 106;
      _np.execute_round_done(ITSIZE);
      // FP: "106 -> 107;
      __syncthreads();
    }
    // FP: "108 -> 109;
    assert(threadIdx.x < __kernel_tb_size);
  }
  // FP: "112 -> 113;
  active_vertices.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(active_vertices_ts);
  // FP: "113 -> 114;
}
__global__ void KCoreSanityCheck(CSRGraph graph, unsigned int __begin, unsigned int __end, uint8_t * p_flag, HGAccumulator<uint64_t> active_vertices)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<uint64_t, TB_SIZE>::TempStorage active_vertices_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  active_vertices.thread_entry();
  // FP: "3 -> 4;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      if (p_flag[src])
      {
        active_vertices.reduce( 1);
      }
    }
  }
  // FP: "11 -> 12;
  active_vertices.thread_exit<hipcub::BlockReduce<uint64_t, TB_SIZE> >(active_vertices_ts);
  // FP: "12 -> 13;
}
void InitializeGraph2_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph2 <<<blocks, __tb_InitializeGraph2>>>(ctx->gg, __begin, __end, ctx->current_degree.data.gpu_wr_ptr(), *(ctx->current_degree.is_updated.gpu_rd_ptr()), t_work.thread_work_wl, t_work.thread_src_wl, enable_lb);
  hipDeviceSynchronize();
  if (enable_lb)
  {
    int num_items = t_work.thread_work_wl.in_wl().nitems();
    if (num_items != 0)
    {
      t_work.compute_prefix_sum();
      hipDeviceSynchronize();
      InitializeGraph2_TB_LB <<<blocks, __tb_InitializeGraph2>>>(ctx->gg, __begin, __end, ctx->current_degree.data.gpu_wr_ptr(), *(ctx->current_degree.is_updated.gpu_rd_ptr()), t_work.thread_prefix_work_wl.gpu_wr_ptr(), num_items, t_work.thread_src_wl);
      hipDeviceSynchronize();
      t_work.reset_thread_work();
    }
  }
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph2_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph2_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph2_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph2_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph2_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph2_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph1_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  t_work.init_thread_work(ctx->gg.nnodes);
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph1 <<<blocks, threads>>>(ctx->gg, __begin, __end, ctx->current_degree.data.gpu_wr_ptr(), ctx->flag.data.gpu_wr_ptr(), ctx->trim.data.gpu_wr_ptr());
  hipDeviceSynchronize();
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph1_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph1_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph1_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph1_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph1_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph1_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void KCoreStep2_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  KCoreStep2 <<<blocks, threads>>>(ctx->gg, __begin, __end, ctx->current_degree.data.gpu_wr_ptr(), ctx->flag.data.gpu_wr_ptr(), ctx->trim.data.gpu_wr_ptr());
  hipDeviceSynchronize();
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void KCoreStep2_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreStep2_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void KCoreStep2_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreStep2_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void KCoreStep2_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreStep2_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void KCoreStep1_cuda(unsigned int  __begin, unsigned int  __end, unsigned int & active_vertices, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<unsigned int> _active_vertices;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<unsigned int> active_verticesval  = Shared<unsigned int>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(active_verticesval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _active_vertices.rv = active_verticesval.gpu_wr_ptr();
  // FP: "8 -> 9;
  KCoreStep1 <<<blocks, __tb_KCoreStep1>>>(ctx->gg, __begin, __end, local_k_core_num, ctx->current_degree.data.gpu_wr_ptr(), ctx->flag.data.gpu_wr_ptr(), ctx->trim.data.gpu_wr_ptr(), *(ctx->trim.is_updated.gpu_rd_ptr()), _active_vertices, t_work.thread_work_wl, t_work.thread_src_wl, enable_lb);
  hipDeviceSynchronize();
  if (enable_lb)
  {
    int num_items = t_work.thread_work_wl.in_wl().nitems();
    if (num_items != 0)
    {
      t_work.compute_prefix_sum();
      hipDeviceSynchronize();
      KCoreStep1_TB_LB <<<blocks, __tb_KCoreStep1>>>(ctx->gg, __begin, __end, local_k_core_num, ctx->current_degree.data.gpu_wr_ptr(), ctx->flag.data.gpu_wr_ptr(), ctx->trim.data.gpu_wr_ptr(), *(ctx->trim.is_updated.gpu_rd_ptr()), _active_vertices, t_work.thread_prefix_work_wl.gpu_wr_ptr(), num_items, t_work.thread_src_wl);
      hipDeviceSynchronize();
      t_work.reset_thread_work();
    }
  }
  // FP: "9 -> 10;
  check_cuda_kernel;
  // FP: "10 -> 11;
  active_vertices = *(active_verticesval.cpu_rd_ptr());
  // FP: "11 -> 12;
}
void KCoreStep1_allNodes_cuda(unsigned int & active_vertices, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreStep1_cuda(0, ctx->gg.nnodes, active_vertices, local_k_core_num, ctx);
  // FP: "2 -> 3;
}
void KCoreStep1_masterNodes_cuda(unsigned int & active_vertices, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreStep1_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, active_vertices, local_k_core_num, ctx);
  // FP: "2 -> 3;
}
void KCoreStep1_nodesWithEdges_cuda(unsigned int & active_vertices, uint32_t local_k_core_num, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreStep1_cuda(0, ctx->numNodesWithEdges, active_vertices, local_k_core_num, ctx);
  // FP: "2 -> 3;
}
void KCoreSanityCheck_cuda(unsigned int  __begin, unsigned int  __end, uint64_t & active_vertices, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<uint64_t> _active_vertices;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<uint64_t> active_verticesval  = Shared<uint64_t>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(active_verticesval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _active_vertices.rv = active_verticesval.gpu_wr_ptr();
  // FP: "8 -> 9;
  KCoreSanityCheck <<<blocks, threads>>>(ctx->gg, __begin, __end, ctx->flag.data.gpu_wr_ptr(), _active_vertices);
  hipDeviceSynchronize();
  // FP: "9 -> 10;
  check_cuda_kernel;
  // FP: "10 -> 11;
  active_vertices = *(active_verticesval.cpu_rd_ptr());
  // FP: "11 -> 12;
}
void KCoreSanityCheck_allNodes_cuda(uint64_t & active_vertices, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreSanityCheck_cuda(0, ctx->gg.nnodes, active_vertices, ctx);
  // FP: "2 -> 3;
}
void KCoreSanityCheck_masterNodes_cuda(uint64_t & active_vertices, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreSanityCheck_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, active_vertices, ctx);
  // FP: "2 -> 3;
}
void KCoreSanityCheck_nodesWithEdges_cuda(uint64_t & active_vertices, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  KCoreSanityCheck_cuda(0, ctx->numNodesWithEdges, active_vertices, ctx);
  // FP: "2 -> 3;
}