#include "hip/hip_runtime.h"
#pragma once
#include "cutils.h"

// flattern data into 1D before feed into the ReLU operater
__global__ void relu_gpu(const int n, const float_t* in, float_t* out) {
	CUDA_KERNEL_LOOP(index, n) {
		out[index] = in[index] > 0 ? in[index] : 0;
	}
}

__global__ void d_relu_gpu(const int n, const float_t* in_diff, const float_t* in_data, float_t* out_diff) {
	CUDA_KERNEL_LOOP(index, n) {
		out_diff[index] = in_data[index] > 0 ? in_diff[index] : 0;
	}
}

void sgemm_gpu(const CBLAS_TRANSPOSE TransA, const CBLAS_TRANSPOSE TransB, 
	const int M, const int N, const int K, const float alpha, 
	const float* A, const float* B, const float beta, float* C) {
	// Note that cublas follows fortran order.
	int lda = (TransA == CblasNoTrans) ? K : M;
	int ldb = (TransB == CblasNoTrans) ? N : K;
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	CUBLAS_CHECK(hipblasSgemm(cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

void gemv_gpu(const CBLAS_TRANSPOSE TransA, const int M, const int N, 
	const float alpha, const float* A, const float* x, const float beta, float* y) {
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	CUBLAS_CHECK(hipblasSgemv(cublas_handle(), cuTransA, N, M, &alpha, A, N, x, 1, &beta, y, 1));
}

void scal_gpu(const int N, const float alpha, float *X) {
	CUBLAS_CHECK(hipblasSscal(cublas_handle(), N, &alpha, X, 1));
}

void dot_gpu(const int n, const float* x, const float* y, float* out) {
	CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

void asum_gpu(const int n, const float* x, float* y) {
	CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

void scale_gpu(const int n, const float alpha, const float *x, float* y) {
	CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
	CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

__global__ void set_kernel(const int n, const float_t alpha, float_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = alpha;
	}
}

void set_gpu(const int N, const float_t alpha, float_t* Y) {
	if (alpha == 0) {
		CUDA_CHECK(hipMemset(Y, 0, sizeof(float_t) * N));
		return;
	}
	set_kernel<float_t><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, alpha, Y);
}

__global__ void add_scalar_kernel(const int n, const float_t alpha, float_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] += alpha;
	}
}

void add_scalar_gpu(const int N, const float alpha, float* Y) {
	add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, alpha, Y);
}

__global__ void add_kernel(const int n, const float_t* a, const float_t* b, float_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = a[index] + b[index];
	}
}

void add_gpu<float>(const int N, const float* a, const float* b, float* y) {
	add_kernel<<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, a, b, y);
}

