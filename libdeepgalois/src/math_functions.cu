#include "hip/hip_runtime.h"
#include "math_functions.hh"
#include "context.h"

void gpu_rng_uniform(const int n, unsigned *r) {
	CURAND_CHECK(hiprandGenerate(Context::curand_generator(), r, n));
}

void gpu_rng_uniform(const int n, const float_t a, const float_t b, float_t* r) {
	CURAND_CHECK(hiprandGenerateUniform(Context::curand_generator(), r, n));
	const float range = b - a;
	if (range != float_t{1}) scal_gpu(n, range, r);
	if (a != float_t{0}) add_scalar_gpu(n, a, r);
}

void gpu_rng_gaussian(const int n, const float_t mu, const float_t sigma, float_t *r) {
	CURAND_CHECK(hiprandGenerateNormal(Context::curand_generator(), r, n, mu, sigma));
}


void gconv_malloc_device(size_t x, size_t y, size_t z, bool dropout, unsigned *masks, float_t *in, float_t *out, float_t *matrix, float_t *grad) {
	if (dropout) CUDA_CHECK(hipMalloc((void **)&masks, x * y * sizeof(unsigned)));
	CUDA_CHECK(hipMalloc((void **)&in, x * y * sizeof(float_t)));
	CUDA_CHECK(hipMalloc((void **)&out, x * z * sizeof(float_t)));
	CUDA_CHECK(hipMalloc((void **)&matrix, y * z * sizeof(float_t)));
	auto init_range = sqrt(6.0/(y + z));
	// Glorot & Bengio (AISTATS 2010)
	gpu_rng_uniform(y*z, -init_range, init_range, matrix);
	CUDA_CHECK(hipMalloc((void **)&grad, y * z * sizeof(float_t)));
	CUDA_CHECK(hipMemset(grad, 0, y * z * sizeof(float_t)));
}

void copy_gpu(size_t len, const float_t *in, float_t *out) {
	CUDA_CHECK(hipMemcpy(out, in, len * sizeof(float_t), hipMemcpyDeviceToDevice));
}

__global__ void dropout_kernel(const int n, const float scale, const float dropout_rate, const float_t* in, unsigned *masks, float_t* out) {
	CUDA_KERNEL_LOOP(i, n) {
		//masks[i] = bernoulli(dropout_rate);
		out[i] = in[i] * masks[i] * scale;
	}
}

void dropout_gpu(const int n, const float scale, const float dropout_rate, const float_t *in, unsigned *masks, float_t *out) {
	dropout_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, scale, dropout_rate, in, masks, out);
}

// flattern data into 1D before feed into the ReLU operater
__global__ void relu_kernel(const int n, const float_t* in, float_t* out) {
	CUDA_KERNEL_LOOP(index, n) {
		out[index] = in[index] > 0 ? in[index] : 0;
	}
}

void relu_gpu(const int n, const float_t *in, float_t* out) {
	relu_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, in, out);
}

__global__ void d_relu_kernel(const int n, const float_t* in_diff, const float_t* data, float_t* out_diff) {
	CUDA_KERNEL_LOOP(index, n) {
		out_diff[index] = data[index] > 0 ? in_diff[index] : 0;
	}
}

void d_relu_gpu(const int n, const float_t *in_diff, const float_t *data, float_t *out_diff) {
	d_relu_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, in_diff, data, out_diff);
}

void sgemm_gpu(const CBLAS_TRANSPOSE TransA, const CBLAS_TRANSPOSE TransB, 
	const int M, const int N, const int K, const float alpha, 
	const float* A, const float* B, const float beta, float* C) {
	// Note that cublas follows fortran order.
	int lda = (TransA == CblasNoTrans) ? K : M;
	int ldb = (TransB == CblasNoTrans) ? N : K;
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	CUBLAS_CHECK(hipblasSgemm(Context::cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

void matmul1D1D_gpu(const size_t dim_x, const size_t dim_y, const size_t dim_z, const float_t *A, const float_t *B, float_t *C) {
	const CBLAS_TRANSPOSE TransA = CblasNoTrans;
	const CBLAS_TRANSPOSE TransB = CblasNoTrans;
	sgemm_gpu(TransA, TransB, dim_x, dim_y, dim_z, 1.0, A, B, 0.0, C);
}

// the arguments of the maxima
int argmax_gpu(const size_t n, const float_t *x) {
	return 0;
}

void gemv_gpu(const CBLAS_TRANSPOSE TransA, const int M, const int N, 
	const float alpha, const float* A, const float* x, const float beta, float* y) {
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	CUBLAS_CHECK(hipblasSgemv(Context::cublas_handle(), cuTransA, N, M, &alpha, A, N, x, 1, &beta, y, 1));
}

void scal_gpu(const int N, const float alpha, float *X) {
	CUBLAS_CHECK(hipblasSscal(Context::cublas_handle(), N, &alpha, X, 1));
}

void dot_gpu(const int n, const float* x, const float* y, float* out) {
	CUBLAS_CHECK(hipblasSdot(Context::cublas_handle(), n, x, 1, y, 1, out));
}

void asum_gpu(const int n, const float* x, float* y) {
	CUBLAS_CHECK(hipblasSasum(Context::cublas_handle(), n, x, 1, y));
}

void scale_gpu(const int n, const float alpha, const float *x, float* y) {
	CUBLAS_CHECK(hipblasScopy(Context::cublas_handle(), n, x, 1, y, 1));
	CUBLAS_CHECK(hipblasSscal(Context::cublas_handle(), n, &alpha, y, 1));
}

__global__ void set_kernel(const int n, const float_t alpha, float_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = alpha;
	}
}

void set_gpu(const int N, const float_t alpha, float_t* Y) {
	if (alpha == 0) {
		CUDA_CHECK(hipMemset(Y, 0, sizeof(float_t) * N));
		return;
	}
	set_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, alpha, Y);
}

__global__ void add_scalar_kernel(const int n, const float_t alpha, float_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] += alpha;
	}
}

void add_scalar_gpu(const int N, const float_t alpha, float_t* Y) {
	add_scalar_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, alpha, Y);
}

__global__ void vadd_kernel(const int n, const float_t* a, const float_t* b, float_t* y) {
	CUDA_KERNEL_LOOP(index, n) {
		y[index] = a[index] + b[index];
	}
}

void vadd_gpu(const int N, const float_t* a, const float_t* b, float_t* y) {
	vadd_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, a, b, y);
}

void softmax_cross_entropy_gpu(int x, int y, const float_t *in_data, float_t *out_data) {
}
