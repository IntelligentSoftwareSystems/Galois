#include "hip/hip_runtime.h"
#include "math_functions.hh"
#include "context.h"
#include "gg.h"
#include "ggcuda.h"
#include "hipcub/hipcub.hpp"
#include <hiprand/hiprand_kernel.h>

__global__ void init_const_kernel(int n, float_t value, float_t *array) {
  CUDA_KERNEL_LOOP(i, n) { array[i] = value; }
}

void init_const_gpu(int n, float_t value, float_t *array) {
  init_const_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, value, array);
  CudaTest("solving init_const kernel failed");
}

__global__ void isnan_test(const int n, const float *data, bool *result) {
	CUDA_KERNEL_LOOP(i, n) { if (isnan(data[i])) *result = true; }
}

bool isnan_gpu(int n, const float_t *array) {
  bool  *d_result, h_result = false;
  hipMalloc((void **)&d_result, sizeof (bool));
  hipMemcpy(d_result, &h_result, sizeof(bool), hipMemcpyHostToDevice);
  isnan_test<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, array, d_result);
  CudaTest("solving init_const kernel failed");
  hipMemcpy(&h_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);
  return h_result;
}

void gpu_rng_uniform(const int n, unsigned* r) {
  CURAND_CHECK(hiprandGenerate(Context::curand_generator(), r, n));
}

void gpu_rng_uniform(const int n, const float_t a, const float_t b, float_t* r) {
  CURAND_CHECK(hiprandGenerateUniform(Context::curand_generator(), r, n));
  const float range = b - a;
  if (range != float_t(1))
    scal_gpu(n, range, r);
  if (a != float_t(0))
    add_scalar_gpu(n, a, r);
}

void gpu_rng_gaussian(const int n, const float_t mu, const float_t sigma, float_t* r) {
  CURAND_CHECK(hiprandGenerateNormal(Context::curand_generator(), r, n, mu, sigma));
}

bool is_allocated_device(float_t* data) {
  if (data == NULL) return false;
  hipPointerAttribute_t attributes;
  CUDA_CHECK(hipPointerGetAttributes(&attributes, data));
  if (attributes.devicePointer != NULL) return true;
  return false;
}

void float_malloc_device(int n, float_t*& loss) {
  CUDA_CHECK(hipMalloc((void**)&loss, n * sizeof(float_t)));
}

void copy_masks_device(int n, mask_t* h_masks, mask_t*& d_masks) {
  assert(h_masks != NULL);
  CUDA_CHECK(hipMalloc((void**)&d_masks, n * sizeof(mask_t)));
  CUDA_CHECK(hipMemcpy(d_masks, h_masks, n * sizeof(mask_t), hipMemcpyHostToDevice));
}

void gconv_malloc_device(size_t x, size_t y, size_t z, bool dropout,
                         unsigned*& masks, float_t*& in, float_t*& out,
                         float_t*& matrix, float_t*& grad) {
  if (dropout) CUDA_CHECK(hipMalloc((void**)&masks, x * y * sizeof(unsigned)));
  CUDA_CHECK(hipMalloc((void**)&in, x * y * sizeof(float_t)));
  init_const_gpu(x*y, 0.0, in);
  CUDA_CHECK(hipMalloc((void**)&out, x * z * sizeof(float_t)));
  init_const_gpu(x*z, 0.0, out);
  CUDA_CHECK(hipMalloc((void**)&matrix, y * z * sizeof(float_t)));
  auto init_range = sqrt(6.0 / (y + z));
  // Glorot & Bengio (AISTATS 2010)
  gpu_rng_uniform(y * z, -init_range, init_range, matrix);
  CUDA_CHECK(hipMalloc((void**)&grad, y * z * sizeof(float_t)));
  CUDA_CHECK(hipMemset(grad, 0, y * z * sizeof(float_t)));
}

__global__ void setup_curand_kernel(const int n, hiprandState* state) {
  CUDA_KERNEL_LOOP(i, n) {
    // hiprand_init(1234, i, 0, &state[i]); // Each thread gets same seed 1234
    hiprand_init(7 + i, i, 0, &state[i]); // Each thread gets different seed
  }
}

__global__ void dropout_kernel(const int n, const float scale,
                               const float threshold, const float_t* in,
                               unsigned* masks, float_t* out) {
  CUDA_KERNEL_LOOP(i, n) { out[i] = in[i] * (masks[i] > threshold) * scale; }
}

void dropout_gpu(const int n, const float scale, const float dropout_rate,
                 const float_t* in, unsigned* masks, float_t* out) {
  gpu_rng_uniform(n, masks);
  //std::cout << "[debug]: dropout_gpu\n";
  dropout_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, scale, dropout_rate, in, masks, out);
  CudaTest("solving dropout kernel failed");
  //std::cout << "[debug]: dropout_gpu done\n";
}

__global__ void d_dropout_kernel(const int n, const float scale,
                                 const float threshold, const float_t* in,
                                 const unsigned* masks, float_t* out) {
  CUDA_KERNEL_LOOP(i, n) { out[i] = in[i] * (masks[i] > threshold) * scale; }
}

void d_dropout_gpu(const int n, const float scale, const float dropout_rate, 
                   const float_t* in, const unsigned* masks, float_t* out) {
  d_dropout_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, scale, dropout_rate, in, masks, out);
  CudaTest("solving d_dropout kernel failed");
}

// flattern data into 1D before feed into the ReLU operater
__global__ void relu_kernel(const int n, const float_t* in, float_t* out) {
  CUDA_KERNEL_LOOP(index, n) { out[index] = in[index] > 0 ? in[index] : 0; }
}

void relu_gpu(const int n, const float_t* in, float_t* out) {
  relu_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, in, out);
  CudaTest("solving relu kernel failed");
}

__global__ void d_relu_kernel(const int n, const float_t* in_diff,
                              const float_t* data, float_t* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = data[index] > 0 ? in_diff[index] : 0;
  }
}

void d_relu_gpu(const int n, const float_t* in_diff, const float_t* data,
                float_t* out_diff) {
  d_relu_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, in_diff, data,
                                                          out_diff);
  CudaTest("solving d_relu kernel failed");
}

__global__ void matmul_kernel(int x, int y, int z, const float_t* A,
                              const float_t* B, float_t* C) {
	int row = blockIdx.x*blockDim.x+threadIdx.x;
	int col = blockIdx.y*blockDim.y+threadIdx.y;
	float_t sum = 0.0f;
	if (row < x && col < y) {
		for (int i = 0; i < z; i++) {
			sum += A[row * z + i] * B[i * y + col];
		}
	}
	C[row * y + col] = sum;
}

#define TILE_SZ 16
void matmul_gpu(const size_t x, const size_t y, const size_t z,
                    const float_t* A, const float_t* B, float_t* C) {
  dim3 threadsPerBlock(TILE_SZ, TILE_SZ);
  dim3 blocksPerGrid((y-1)/TILE_SZ+1, (x-1)/TILE_SZ+1);
  matmul_kernel<<<blocksPerGrid,threadsPerBlock>>>(x, y, z, A, B, C);
  CudaTest("solving matmul kernel failed");
}

void sgemm_gpu(const CBLAS_TRANSPOSE TransA, const CBLAS_TRANSPOSE TransB,
               const int M, const int N, const int K, const float alpha,
               const float* A, const float* B, const float beta, float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Context::cublas_handle(), cuTransB, cuTransA,
                           N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

void matmul1D1D_gpu(const size_t dim_x, const size_t dim_y, const size_t dim_z,
                    const float_t* A, const float_t* B, float_t* C) {
  // std::cout << "[debug]: matmul1D1D_gpu\n";
  const CBLAS_TRANSPOSE TransA = CblasNoTrans;
  const CBLAS_TRANSPOSE TransB = CblasNoTrans;
  sgemm_gpu(TransA, TransB, dim_x, dim_y, dim_z, 1.0, A, B, 0.0, C);
}

void gemv_gpu(const CBLAS_TRANSPOSE TransA, const int M, const int N,
              const float alpha, const float* A, const float* x,
              const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Context::cublas_handle(), cuTransA, N, M, &alpha, A,
                           N, x, 1, &beta, y, 1));
}

void scal_gpu(const int N, const float alpha, float* X) {
  CUBLAS_CHECK(hipblasSscal(Context::cublas_handle(), N, &alpha, X, 1));
}

void dot_gpu(const int n, const float* x, const float* y, float* out) {
  CUBLAS_CHECK(hipblasSdot(Context::cublas_handle(), n, x, 1, y, 1, out));
}

void asum_gpu(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Context::cublas_handle(), n, x, 1, y));
}

void scale_gpu(const int n, const float alpha, const float* x, float* y) {
  CUBLAS_CHECK(hipblasScopy(Context::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Context::cublas_handle(), n, &alpha, y, 1));
}

__global__ void set_kernel(const int n, const float_t alpha, float_t* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = alpha; }
}

void set_gpu(const int N, const float_t alpha, float_t* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(float_t) * N));
    return;
  }
  set_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, alpha, Y);
  CudaTest("solving set kernel failed");
}

__global__ void add_scalar_kernel(const int n, const float_t alpha,
                                  float_t* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] += alpha; }
}

void add_scalar_gpu(const int N, const float_t alpha, float_t* Y) {
  add_scalar_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, alpha, Y);
  CudaTest("solving add_scalar kernel failed");
}

__global__ void vadd_kernel(const int n, const float_t* a, const float_t* b,
                            float_t* y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = a[index] + b[index]; }
}

void copy_gpu(int len, const float_t* in, float_t* out) {
  CUDA_CHECK(hipMemcpy(out, in, len * sizeof(float_t), hipMemcpyDeviceToDevice));
}

void vadd_gpu(const int N, const float_t* a, const float_t* b, float_t* y) {
  vadd_kernel<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, a, b, y);
  CudaTest("solving vadd kernel failed");
}

// TODO: use warp
__device__ void softmax_device(int n, const float_t* input, float_t* output) {
  float_t max = input[0];
  for (int i = 1; i < n; i++)
    if (input[i] > max)
      max = input[i];
  float_t denominator = 0.0;
  for (int i = 0; i < n; i++) {
    output[i] = expf(input[i] - max);
    denominator += output[i];
	if (output[i] < 0.0) printf("in[%d]=%f, out[%d]=%f\n", i, input[i], i, output[i]);
    //assert(output[i] >= 0.0);
  }
  assert(denominator != 0.0);
  for (int i = 0; i < n; i++) {
    output[i] /= denominator;
    //assert(output[i] >= 0.0);
    //assert(output[i] <= 1.0);
  }
}

__device__ void cross_entropy_device(int n, const label_t idx, const float_t* p, float_t& loss) {
  if (p[idx] == 0.0) loss -= logf(float_t(1e-10));
  else loss -= logf(p[idx]);
}

// n: number of vectors
// len: length of vectors
// for each vector, do softmax to normalize the vector, and then compute a loss
__global__ void softmax_cross_entropy_kernel(int len, int begin, int end,
                                             const float_t* in_data,
                                             const mask_t* masks,
                                             const label_t* labels,
                                             float_t* loss, float_t* out_data) {
  CUDA_KERNEL_LOOP(i, end-begin) {
    int id = begin + i;
    if (masks[id] == 1) { // masked
	  // normalize using softmax
      softmax_device(len, in_data + len*id, out_data + len*id);
      //loss[id] = 0.0;
      cross_entropy_device(len, labels[id], out_data + len*id, loss[id]);
    }
  }
}

void softmax_cross_entropy_gpu(int len, int begin, int end, const float_t* in,
                               const mask_t* masks, const label_t* labels,
                               float_t* loss, float_t* out) {
  softmax_cross_entropy_kernel<<<CUDA_GET_BLOCKS(end-begin), CUDA_NUM_THREADS>>>(
      len, begin, end, in, masks, labels, loss, out);
  CudaTest("solving softmax_cross_entropy kernel failed");
}

// TODO: use warp
__device__ void d_softmax(int n, const float_t* p, const float_t* dp, float_t* dy) {
  for (int i = 0; i < n; i++) {
    dy[i] = 0;
    for (int j = 0; j < n; j++) {
      float_t df = (j == i) ? p[i] * (1.0 - p[i]) : -p[j] * p[i];
      dy[i] += df * dp[j];
    }
  }
}

__device__ void d_cross_entropy(int n, const label_t idx, const float_t* p, float_t* d) {
  for (int i = 0; i < n; i++) {
    //assert(p[i] >= 0.0);
    //assert(p[i] >= 0.0 && p[i] <= 1.0);
    if (i == (int)idx) d[i] = -1.0 / (p[i] + 1e-10);
    else d[i] = 0.0;
  }
}

__global__ void d_softmax_cross_entropy_kernel(int len, int begin, int end,
                               const mask_t* masks, const label_t* labels,
                               const float_t* out, float_t* diff) {
  CUDA_KERNEL_LOOP(i, end-begin) {
    int id = begin + i;
    if (masks[id] == 1) { // masked
	  float_t out_grad[41]; // TODO
      d_cross_entropy(len, labels[id], out + len*id, out_grad);
      d_softmax(len, out + len*id, out_grad, diff + len*id);
    }
  }
}

void d_softmax_cross_entropy_gpu(int len, int begin, int end,
                                 const mask_t* masks, const label_t* labels,
                                 const float_t* out, float_t* diff) {
  d_softmax_cross_entropy_kernel<<<CUDA_GET_BLOCKS(end-begin), CUDA_NUM_THREADS>>>(
      len, begin, end, masks, labels, out, diff);
  CudaTest("solving d_softmax_cross_entropy kernel failed");
}

__global__ void masked_avg_loss_kernel(int begin, int end, mask_t* masks,
                                       float_t* loss,
                                       HGAccumulator<acc_t> total) {
  total.thread_entry();
  __shared__ hipcub::BlockReduce<acc_t, CUDA_NUM_THREADS>::TempStorage local_loss;
  CUDA_KERNEL_LOOP(i, end - begin) {
    if (masks[begin + i] == 1)
      // total += loss[begin+i];
      total.reduce(loss[begin + i]);
  }
  total.thread_exit<hipcub::BlockReduce<acc_t, CUDA_NUM_THREADS>>(local_loss);
}

acc_t masked_avg_loss(int begin, int end, int count, mask_t* masks,
                      float_t* loss) {
  assert(count > 0);
  HGAccumulator<acc_t> loss_accum;
  Shared<acc_t> total_loss   = Shared<acc_t>(1);
  *(total_loss.cpu_wr_ptr()) = 0;
  loss_accum.rv              = total_loss.gpu_wr_ptr();
  masked_avg_loss_kernel<<<CUDA_GET_BLOCKS(end - begin), CUDA_NUM_THREADS>>>(
      begin, end, masks, loss, loss_accum);
  CudaTest("solving masked_avg_loss kernel failed");
  hipDeviceSynchronize();
  return *(total_loss.cpu_rd_ptr()) / count;
}

// the arguments of the maxima
__device__ int argmax_device(const int n, const float_t* x) {
  float_t max    = x[0];
  int max_ind = 0;
  for (int i = 1; i < n; i++) {
    if (x[i] > max) {
      max_ind = i;
      max     = x[i];
    }
  }
  return max_ind;
}

__global__ void masked_accuracy_kernel(int num_classes, int begin,
                                       int end, mask_t* masks,
                                       float_t* preds, label_t* labels,
                                       HGAccumulator<acc_t> total) {
  total.thread_entry();
  __shared__ hipcub::BlockReduce<acc_t, CUDA_NUM_THREADS>::TempStorage
      local_accuracy;
  CUDA_KERNEL_LOOP(i, end - begin) {
    if (masks[begin + i] == 1) {
      label_t pred = (label_t)argmax_device(num_classes,
                                            preds + (begin + i) * num_classes);
      if (pred == labels[begin + i])
        total.reduce(1.0);
    }
  }
  total.thread_exit<hipcub::BlockReduce<acc_t, CUDA_NUM_THREADS>>(local_accuracy);
}

acc_t masked_accuracy_gpu(int num_classes, int begin, int end,
                          int count, mask_t* masks, float_t* preds,
                          label_t* labels) {
  assert(count > 0);
  HGAccumulator<acc_t> accuracy_accum;
  Shared<acc_t> total_accuracy   = Shared<acc_t>(1);
  *(total_accuracy.cpu_wr_ptr()) = 0;
  accuracy_accum.rv              = total_accuracy.gpu_wr_ptr();
  masked_accuracy_kernel<<<CUDA_GET_BLOCKS(end - begin), CUDA_NUM_THREADS>>>(
      num_classes, begin, end, masks, preds, labels, accuracy_accum);
  CudaTest("solving masked_avg_loss kernel failed");
  hipDeviceSynchronize();
  return *(total_accuracy.cpu_rd_ptr()) / count;
}
