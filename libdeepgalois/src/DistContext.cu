#include "hip/hip_runtime.h"
#include <ctime>
#include <cstdio>
#include <unistd.h>
#include <sys/types.h>
#include "deepgalois/DistContext.h"
#include "deepgalois/math_functions.hh"
#include "deepgalois/configs.h"

// random seeding
int64_t cluster_seedgen(void) {
  int64_t s, seed, pid;
  FILE* f = fopen("/dev/urandom", "rb");
  if (f && fread(&seed, 1, sizeof(seed), f) == sizeof(seed)) {
    fclose(f);
    return seed;
  }
  std::cout << "System entropy source not available, "
               "using fallback algorithm to generate seed instead.";
  if (f)
    fclose(f);
  pid  = getpid();
  s    = time(NULL);
  seed = std::abs(((s * 181) * ((pid - 83) * 359)) % 104729);
  return seed;
}

namespace deepgalois {

// computing normalization factor for each vertex
__global__ void norm_factor_computing_node(int n, GraphGPU graph, float_t* norm_fac) {
  CUDA_KERNEL_LOOP(i, n) {
    float_t temp = sqrt(float_t(graph.getOutDegree(i)));
    if (temp == 0.0) norm_fac[i] = 0.0;
    else norm_fac[i] = 1.0 / temp;
  }
}

// TODO: make sure self-loop added for each vertex
// computing normalization factor for each edge
__global__ void norm_factor_computing_edge(int n, GraphGPU graph, float_t* norm_fac) {
  CUDA_KERNEL_LOOP(src, n) {
    assert(src < n);
    float_t d_src = float_t(graph.getOutDegree(src));
    assert(d_src != 0.0); // should never be zero since self-loop added for each vertex
    d_src       = 1.0 / sqrt(d_src);
    auto start  = graph.edge_begin(src);
    index_t end = graph.edge_end(src);
    for (index_t e = start; e != end; e++) {
      index_t dst = graph.getEdgeDst(e);
      // if (dst >= n) printf("src=%d, dst=%d, e=%d, start=%d, end=%d\n", src,
      // dst, e, start, end);
      assert(dst < n);
      float_t d_dst = float_t(graph.getOutDegree(dst));
      assert(d_dst != 0.0);
      d_dst       = 1.0 / sqrt(d_dst);
      norm_fac[e] = d_src * d_dst;
    }
  }
}

hipblasHandle_t DistContext::cublas_handle_         = 0;
hipsparseHandle_t DistContext::cusparse_handle_     = 0;
hipsparseMatDescr_t DistContext::cusparse_matdescr_ = 0;
hiprandGenerator_t DistContext::curand_generator_   = 0;

DistContext::DistContext() : DistContext(true) {
  d_labels = NULL; 
  d_feats = NULL;
  d_labels_subg = NULL; 
  d_feats_subg = NULL;
  d_normFactors = NULL;
  d_normFactorsSub = NULL;
  CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
  CUSPARSE_CHECK(hipsparseCreate(&cusparse_handle_));
  CUSPARSE_CHECK(hipsparseCreateMatDescr(&cusparse_matdescr_));
  CUSPARSE_CHECK(
      hipsparseSetMatType(cusparse_matdescr_, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CHECK(
      hipsparseSetMatIndexBase(cusparse_matdescr_, HIPSPARSE_INDEX_BASE_ZERO));
  CURAND_CHECK(
      hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CHECK(
      hiprandSetPseudoRandomGeneratorSeed(curand_generator_, cluster_seedgen()));
}

DistContext::~DistContext() {
  if (cublas_handle_)
    CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
  if (cusparse_handle_)
    CUSPARSE_CHECK(hipsparseDestroy(cusparse_handle_));
  if (cusparse_matdescr_)
    CUSPARSE_CHECK(hipsparseDestroyMatDescr(cusparse_matdescr_));
  if (curand_generator_)
    CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
  if (d_labels) CUDA_CHECK(hipFree(d_labels));
  if (d_feats) CUDA_CHECK(hipFree(d_feats));
  if (d_normFactors) CUDA_CHECK(hipFree(d_normFactors));
  if (d_labels_subg) CUDA_CHECK(hipFree(d_labels_subg));
  if (d_feats_subg) CUDA_CHECK(hipFree(d_feats_subg));
  if (d_normFactorsSub) CUDA_CHECK(hipFree(d_normFactorsSub));
}

size_t DistContext::read_labels(bool isSingleClass, std::string dataset_str) {
  num_classes = reader.read_labels(isSingleClass, h_labels);
  return num_classes;
}

size_t DistContext::read_features(std::string dataset_str) {
  feat_len = reader.read_features(h_feats);
  return feat_len;
}

size_t DistContext::read_masks(std::string dataset_str, std::string mask_type, size_t n, 
                               size_t& begin, size_t& end, mask_t* masks, DGraph* dGraph) {
  return reader.read_masks(mask_type, n, begin, end, masks);
}

//! allocate memory for subgraphs (don't actually build them)
void DistContext::allocateSubgraphs(int num_subgraphs, unsigned max_size) {
  this->partitionedSubgraphs.resize(num_subgraphs);
  for (int i = 0; i < num_subgraphs; i++) {
    this->partitionedSubgraphs[i] = new Graph();
    this->partitionedSubgraphs[i]->set_max_size(max_size);
  }
}

void DistContext::constructSubgraphLabels(size_t m, const mask_t* masks) {
  size_t labels_size = m;
  if (!usingSingleClass) labels_size = m * num_classes;
  h_labels_subg.resize(labels_size);
  size_t count = 0;
  for (size_t i = 0; i < this->partitionedGraph->size(); i++) {
    if (masks[i] == 1) {
      if (usingSingleClass) h_labels_subg[count] = h_labels[i];
      else std::copy(h_labels + i * num_classes, h_labels + (i + 1) * num_classes, 
                     &h_labels_subg[count * num_classes]);
      count++;
    }
  }
  if (d_labels_subg) uint8_free_device(d_labels_subg);
  uint8_malloc_device(labels_size, d_labels_subg);
  uint8_copy_device(labels_size, &h_labels_subg[0], d_labels_subg);
}

void DistContext::constructSubgraphFeatures(size_t m, const mask_t* masks) {
  //std::cout << "construct subgraph features (d_feats_subg: " << d_feats_subg << ") ... ";
  size_t count = 0;
  DistContext::h_feats_subg.resize(m * feat_len);
  for (size_t i = 0; i < this->partitionedGraph->size(); i++) {
    if (masks[i] == 1) {
      std::copy(h_feats + i * feat_len, h_feats + (i + 1) * feat_len, &h_feats_subg[count * feat_len]);
      count++;
    }
  }
  if (d_feats_subg) float_free_device(d_feats_subg);
  float_malloc_device(m * feat_len, d_feats_subg);
  float_copy_device(m * feat_len, &h_feats_subg[0], d_feats_subg);
  //std::cout << "Done\n";
}

void DistContext::constructNormFactorSub(int subgraphID) {
  Graph& graphToUse = *partitionedSubgraphs[subgraphID];
  auto n = graphToUse.size();
  //std::cout << "Pre-computing subgraph normalization factor (n=" << n << ") ... ";

 #ifdef USE_CUSPARSE
  auto nnz = graphToUse.sizeEdges();
  float_malloc_device(nnz, d_normFactorsSub);
  init_const_gpu(nnz, 0.0, d_normFactors);
  norm_factor_computing_edge<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, graphToUse, d_normFactorsSub);
#else
  float_malloc_device(n, d_normFactorsSub);
  norm_factor_computing_node<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, graphToUse, d_normFactorsSub);
#endif
  CudaTest("solving norm_factor_computing kernel failed");
  //std::cout << "Done\n";
}

void DistContext::constructNormFactor(deepgalois::Context* globalContext) {
  auto n = partitionedGraph->size();
  std::cout << "Pre-computing normalization factor (n=" << n << ") ... ";
  if (!is_selfloop_added) {
    std::cout << "Set -sl=1 to add selfloop\n";
    exit(0);
  }
#ifdef USE_CUSPARSE
  auto nnz = partitionedGraph->sizeEdges();
  CUDA_CHECK(hipMalloc((void**)&d_normFactors, nnz * sizeof(float_t)));
  init_const_gpu(nnz, 0.0, d_normFactors);
  norm_factor_computing_edge<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, *partitionedGraph, d_normFactors);
#else
  CUDA_CHECK(hipMalloc((void**)&d_normFactors, n * sizeof(float_t)));
  norm_factor_computing_node<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, *partitionedGraph, d_normFactors);
#endif
  CudaTest("solving norm_factor_computing kernel failed");
  std::cout << "Done\n";
}

/*
void DistContext::SetDevice(const int device_id) {
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) return;
  CUDA_CHECK(hipSetDevice(device_id));
  if (cublas_handle_) CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
  if (curand_generator_)
CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
  CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
  CURAND_CHECK(hiprandCreateGenerator(&curand_generator_,
HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curand_generator_,
cluster_seedgen()));
}
*/
size_t DistContext::read_graph(std::string dataset, bool selfloop) {
  partitionedGraph = new DGraph();
#ifdef USE_CSRGRAPH
  std::string filename = path + dataset + ".csgr";
  GraphGPU g;
  g.read(filename.c_str(), false);
  if (selfloop) {
    g.add_selfloop();
    is_selfloop_added = selfloop;
  }
  g.copy_to_gpu(*partitionedGraph);
#else
  partitionedGraph->readGraph(dataset);
  if (selfloop) {
    partitionedGraph->add_selfloop();
    is_selfloop_added = selfloop;
  }
  partitionedGraph->copy_to_gpu();
#endif
  return partitionedGraph->size();
}

void DistContext::copy_data_to_device() {
  auto n = partitionedGraph->size();
  std::cout << "Copying labels and features to GPU memory. n = " << n << " ... ";
  if (usingSingleClass) {
    CUDA_CHECK(hipMalloc((void**)&d_labels, n * sizeof(label_t)));
    CUDA_CHECK(hipMemcpy(d_labels, h_labels, n * sizeof(label_t), hipMemcpyHostToDevice));
  } else {
    CUDA_CHECK(hipMalloc((void**)&d_labels, n * num_classes * sizeof(label_t)));
    CUDA_CHECK(hipMemcpy(d_labels, h_labels, n * num_classes * sizeof(label_t), hipMemcpyHostToDevice));
  }
  CUDA_CHECK(hipMalloc((void**)&d_feats, n * feat_len * sizeof(float_t)));
  CUDA_CHECK(hipMemcpy(d_feats, &h_feats[0], n * feat_len * sizeof(float_t), hipMemcpyHostToDevice));
  // print_device_vector(10, d_feats, "d_feats");
  std::cout << "Done\n";
}

} // namespace deepgalois
