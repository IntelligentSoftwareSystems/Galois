#include "hip/hip_runtime.h"
#include <ctime>
#include <cstdio>
#include <unistd.h>
#include <sys/types.h>
#include "deepgalois/context.h"
#include "deepgalois/math_functions.hh"

// random seeding
int64_t cluster_seedgen(void) {
  int64_t s, seed, pid;
  FILE* f = fopen("/dev/urandom", "rb");
  if (f && fread(&seed, 1, sizeof(seed), f) == sizeof(seed)) {
    fclose(f);
    return seed;
  }
  std::cout << "System entropy source not available, "
               "using fallback algorithm to generate seed instead.";
  if (f)
    fclose(f);
  pid  = getpid();
  s    = time(NULL);
  seed = std::abs(((s * 181) * ((pid - 83) * 359)) % 104729);
  return seed;
}

// computing normalization factor for each vertex
__global__ void norm_factor_counting_node(int n, CSRGraph graph, float_t* norm_fac) {
  CUDA_KERNEL_LOOP(i, n) {
    float_t temp = sqrt(float_t(graph.getOutDegree(i)));
    if (temp == 0.0) norm_fac[i] = 0.0;
    else norm_fac[i] = 1.0 / temp;
  }
}

// TODO: make sure self-loop added for each vertex
// computing normalization factor for each edge
__global__ void norm_factor_counting_edge(int n, CSRGraph graph, float_t* norm_fac) {
  CUDA_KERNEL_LOOP(src, n) {
    assert(src < n);
    float_t d_src = float_t(graph.getOutDegree(src));
    assert(d_src != 0.0); // should never be zero since self-loop added for each vertex
    d_src = 1.0 / sqrt(d_src);
    index_type start = graph.edge_begin(src);
    index_type end = graph.edge_end(src);
	for (index_type e = start; e != end; e++) {
      index_type dst = graph.getEdgeDst(e);
      if (dst >= n) printf("src=%d, dst=%d, e=%d, start=%d, end=%d\n", src, dst, e, start, end);
      assert(dst < n);
      float_t d_dst = float_t(graph.getOutDegree(dst));
      assert(d_dst != 0.0);
      d_dst = 1.0 / sqrt(d_dst);
      norm_fac[e] = d_src * d_dst;
    }
  }
}

namespace deepgalois {

hipblasHandle_t Context::cublas_handle_         = 0;
hipsparseHandle_t Context::cusparse_handle_     = 0;
hipsparseMatDescr_t Context::cusparse_matdescr_ = 0;
hiprandGenerator_t Context::curand_generator_   = 0;

Context::Context() : n(0), num_classes(0), feat_len(0), 
  is_single_class(true), is_selfloop_added(false), 
  labels(NULL), h_feats(NULL), norm_factor(NULL),
  d_labels(NULL), d_feats(NULL) {
  CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
  CUSPARSE_CHECK(hipsparseCreate(&cusparse_handle_));
  CUSPARSE_CHECK(hipsparseCreateMatDescr(&cusparse_matdescr_));
  CUSPARSE_CHECK(hipsparseSetMatType(cusparse_matdescr_,HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CHECK(hipsparseSetMatIndexBase(cusparse_matdescr_,HIPSPARSE_INDEX_BASE_ZERO));
  CURAND_CHECK(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, cluster_seedgen()));
}

Context::~Context() {
  if (cublas_handle_)
    CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
  if (cusparse_handle_)
    CUSPARSE_CHECK(hipsparseDestroy(cusparse_handle_));
  if (cusparse_matdescr_)
    CUSPARSE_CHECK(hipsparseDestroyMatDescr(cusparse_matdescr_));
  if (curand_generator_)
    CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
}

size_t Context::read_graph(std::string dataset_str, bool selfloop) {
  n = read_graph_gpu(dataset_str, selfloop);
  return n;
}

void Context::norm_factor_counting() {
  std::cout << "Pre-computing normalization factor (n=" << n << ") ... ";
  if (!is_selfloop_added) {
    std::cout << "Set -sl=1 to add selfloop\n";	  
    exit(0);
  }
#ifdef USE_CUSPARSE
  int nnz = graph_gpu.nedges;
  CUDA_CHECK(hipMalloc((void**)&norm_factor, nnz * sizeof(float_t)));
  init_const_gpu(nnz, 0.0, norm_factor);
  norm_factor_counting_edge<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, graph_gpu, norm_factor);
#else
  CUDA_CHECK(hipMalloc((void**)&norm_factor, n * sizeof(float_t)));
  norm_factor_counting_node<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, graph_gpu, norm_factor);
#endif
  CudaTest("solving norm_factor_counting kernel failed");
  std::cout << "Done\n";
}
/*
void Context::SetDevice(const int device_id) {
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) return;
  CUDA_CHECK(hipSetDevice(device_id));
  if (cublas_handle_) CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
  if (curand_generator_) CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
  CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
  CURAND_CHECK(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, cluster_seedgen()));
}
*/
size_t Context::read_graph_gpu(std::string dataset_str, bool selfloop) {
  std::string filename = path + dataset_str + ".csgr";
  CSRGraph g;
  g.read(filename.c_str(), false);
  if (selfloop) {
    g.add_selfloop();
    is_selfloop_added = selfloop;
  }
  g.copy_to_gpu(graph_gpu);
  return graph_gpu.nnodes;
}

void Context::copy_data_to_device() {
  CUDA_CHECK(hipMalloc((void**)&d_labels, n * sizeof(label_t)));
  CUDA_CHECK(hipMemcpy(d_labels, labels, n * sizeof(label_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc((void**)&d_feats, n * feat_len * sizeof(float_t)));
  CUDA_CHECK(hipMemcpy(d_feats, &h_feats[0], n * feat_len * sizeof(float_t),
                        hipMemcpyHostToDevice));
  //print_device_vector(10, d_feats, "d_feats");
}

//void Context::copy_data_to_device() {
  //float_malloc_device(n, d_labels);
  //float_copy_device(n, labels, d_labels);
  //float_malloc_device(n*feat_len, d_feats);
  //float_copy_device(n*feat_len, &h_feats[0], d_feats);
//}

float_t* Context::get_in_ptr() { return d_feats; }
} // namespace context
