#include "hip/hip_runtime.h"
#include <ctime>
#include <cstdio>
#include <unistd.h>
#include <sys/types.h>
#include "context.h"

// random seeding
int64_t cluster_seedgen(void) {
  int64_t s, seed, pid;
  FILE* f = fopen("/dev/urandom", "rb");
  if (f && fread(&seed, 1, sizeof(seed), f) == sizeof(seed)) {
    fclose(f);
    return seed;
  }
  std::cout << "System entropy source not available, "
               "using fallback algorithm to generate seed instead.";
  if (f)
    fclose(f);
  pid  = getpid();
  s    = time(NULL);
  seed = std::abs(((s * 181) * ((pid - 83) * 359)) % 104729);
  return seed;
}

// computing normalization factor for each vertex
__global__ void norm_factor_counting_node(int n, CSRGraph graph,
                                            float_t* norm_fac) {
  CUDA_KERNEL_LOOP(i, n) {
    float_t temp = sqrt(float_t(graph.getOutDegree(i)));
    if (temp == 0.0)
      norm_fac[i] = 0.0;
    else
      norm_fac[i] = 1.0 / temp;
  }
}

// TODO: make sure self-loop added for each vertex
// computing normalization factor for each edge
__global__ void norm_factor_counting_edge(int n, CSRGraph graph,
                                            float_t* norm_fac) {
  CUDA_KERNEL_LOOP(src, n) {
    float_t d_src = float_t(graph.getOutDegree(src));
    assert(d_src != 0.0); // should never be zero since self-loop added for each vertex
    d_src = 1.0 / sqrt(d_src);
    index_type start = graph.edge_begin(src);
    index_type end = graph.edge_end(src);
	for (index_type e = start; e != end; e++) {
      index_type dst = graph.getEdgeDst(e);
      float_t d_dst = float_t(graph.getOutDegree(dst));
      assert(d_dst != 0.0);
      d_dst = 1.0 / sqrt(d_dst);
      norm_fac[e] = d_src * d_dst;
    }
  }
}

namespace deepgalois {

void Context::norm_factor_counting_gpu() {
  assert(graph_gpu.nnodes == n);
  std::cout << "Pre-computing normalization factor (n=" << n << ")\n";
#ifdef USE_CUSPARSE
  int nnz = graph_gpu.nedges;
  CUDA_CHECK(hipMalloc((void**)&d_norm_factor, nnz * sizeof(float_t)));
  init_const_kernel<<<CUDA_GET_BLOCKS(nnz), CUDA_NUM_THREADS>>>(nnz, 0.0, d_norm_factor);
  norm_factor_counting_edge<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, graph_gpu, d_norm_factor);
#else
  CUDA_CHECK(hipMalloc((void**)&d_norm_factor, n * sizeof(float_t)));
  norm_factor_counting_node<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(
      n, graph_gpu, d_norm_factor);
#endif
  CudaTest("solving norm_factor_counting kernel failed");
}

hipblasHandle_t Context::cublas_handle_         = 0;
hipsparseHandle_t Context::cusparse_handle_     = 0;
hipsparseMatDescr_t Context::cusparse_matdescr_ = 0;
hiprandGenerator_t Context::curand_generator_   = 0;

Context::Context()
    : mode_(Context::GPU), solver_count_(1), solver_rank_(0),
      multiprocess_(false) {
  CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
  CUSPARSE_CHECK(hipsparseCreate(&cusparse_handle_));
  CUSPARSE_CHECK(hipsparseCreateMatDescr(&cusparse_matdescr_));
  CUSPARSE_CHECK(hipsparseSetMatType(cusparse_matdescr_,HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CHECK(hipsparseSetMatIndexBase(cusparse_matdescr_,HIPSPARSE_INDEX_BASE_ZERO));
  CURAND_CHECK(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, cluster_seedgen()));
}

Context::~Context() {
  if (cublas_handle_)
    CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
  if (cusparse_handle_)
    CUSPARSE_CHECK(hipsparseDestroy(cusparse_handle_));
  if (cusparse_matdescr_)
    CUSPARSE_CHECK(hipsparseDestroyMatDescr(cusparse_matdescr_));
  if (curand_generator_)
    CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
}

void Context::SetDevice(const int device_id) {
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id)
    return;
  CUDA_CHECK(hipSetDevice(device_id));
  if (cublas_handle_)
    CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
  if (curand_generator_)
    CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
  CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
  CURAND_CHECK(
      hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CHECK(
      hiprandSetPseudoRandomGeneratorSeed(curand_generator_, cluster_seedgen()));
}

size_t Context::read_graph_gpu(std::string dataset_str, bool selfloop) {
  std::string filename = path + dataset_str + ".csgr";
  CSRGraph g;
  g.read(filename.c_str(), false);
  if (selfloop) g.add_selfloop();
  g.copy_to_gpu(graph_gpu);
  return graph_gpu.nnodes;
}

void Context::copy_data_to_device() {
  assert(labels.size() == n);
  CUDA_CHECK(hipMalloc((void**)&d_labels, n * sizeof(label_t)));
  CUDA_CHECK(hipMemcpy(d_labels, &labels[0], n * sizeof(label_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc((void**)&d_feats, n * feat_len * sizeof(float_t)));
  CUDA_CHECK(hipMemcpy(d_feats, &h_feats[0], n * feat_len * sizeof(float_t),
                        hipMemcpyHostToDevice));
  //print_device_vector(10, d_feats, "d_feats");
}

float_t* Context::get_in_ptr() { return d_feats; }
} // namespace context
