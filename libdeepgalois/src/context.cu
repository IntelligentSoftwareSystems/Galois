#include "hip/hip_runtime.h"
#include <ctime>
#include <cstdio>
#include <unistd.h>
#include <sys/types.h>
#include "context.h"

// random seeding
int64_t cluster_seedgen(void) {
	int64_t s, seed, pid;
	FILE* f = fopen("/dev/urandom", "rb");
	if (f && fread(&seed, 1, sizeof(seed), f) == sizeof(seed)) {
		fclose(f);
		return seed;
	}
	std::cout << "System entropy source not available, "
		"using fallback algorithm to generate seed instead.";
	if (f) fclose(f);
	pid = getpid();
	s = time(NULL);
	seed = std::abs(((s * 181) * ((pid - 83) * 359)) % 104729);
	return seed;
}

__global__ void norm_factor_counting_kernel(size_t n, CSRGraph graph, float_t *norm_factor) {
	CUDA_KERNEL_LOOP(i, n) {
		float_t temp = sqrt(float_t(graph.getOutDegree(i)));
		if (temp == 0.0) norm_factor[i] = 0.0;
		else norm_factor[i] = 1.0 / temp;
	}
}

void Context::norm_factor_counting_gpu(size_t n, CSRGraph graph, float_t *norm_factor) {
	CUDA_CHECK(hipMalloc((void **)&norm_factor, n * sizeof(float_t)));
	norm_factor_counting_kernel<<<CUDA_GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, graph, norm_factor);
}

hipblasHandle_t Context::cublas_handle_ = 0;
hiprandGenerator_t Context::curand_generator_ = 0;

Context::Context() : mode_(Context::GPU), solver_count_(1), 
	solver_rank_(0), multiprocess_(false) {
	CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
	CURAND_CHECK(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, cluster_seedgen()));
}

Context::~Context() {
	if (cublas_handle_) CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
	if (curand_generator_) {
		CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
	}
}

void Context::SetDevice(const int device_id) {
	int current_device;
	CUDA_CHECK(hipGetDevice(&current_device));
	if (current_device == device_id) return;
	CUDA_CHECK(hipSetDevice(device_id));
	if (cublas_handle_) CUBLAS_CHECK(hipblasDestroy(cublas_handle_));
	if (curand_generator_) CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
	CUBLAS_CHECK(hipblasCreate(&cublas_handle_));
	CURAND_CHECK(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, cluster_seedgen()));
}

size_t Context::read_graph_gpu(std::string dataset_str) {
	std::string filename = path + dataset_str + ".csgr";
	graph_gpu.read(filename.c_str(), false);
	return graph_gpu.nnodes;
}

void Context::copy_data_to_device() {
	CUDA_CHECK(hipMalloc((void **)&d_labels, n * sizeof(label_t)));
	CUDA_CHECK(hipMemcpy(d_labels, &labels[0], n * sizeof(label_t), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMalloc((void **)&d_feats, n * feat_len *  sizeof(float_t)));
	CUDA_CHECK(hipMemcpy(d_feats, &h_feats[0], n * feat_len * sizeof(float_t), hipMemcpyHostToDevice));
}

float_t * Context::get_in_ptr() { return d_feats; }

