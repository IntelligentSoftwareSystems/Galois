#include "hip/hip_runtime.h"
#include "deepgalois/layers/graph_conv_layer.h"

namespace deepgalois {

void graph_conv_layer::init() {
  if (dropout_) CUDA_CHECK(hipMalloc((void**)&dropout_mask, x * y * sizeof(unsigned)));
  //CUDA_CHECK(hipMalloc((void**)&in_temp, x * y * sizeof(float_t)));
  float_malloc_device(x*y, in_temp);
  init_const_gpu(x*y, 0.0, in_temp);
  if (y <= z) {
    float_malloc_device(x*y, in_temp1);
    init_const_gpu(x*y, 0.0, in_temp1);
  }
  //CUDA_CHECK(hipMalloc((void**)&out_temp, x * z * sizeof(float_t)));
  float_malloc_device(x*z, out_temp);
  init_const_gpu(x*z, 0.0, out_temp);
  //CUDA_CHECK(hipMalloc((void**)&d_W, y * z * sizeof(float_t)));
  float_malloc_device(y*z, d_W);
  auto init_range = sqrt(6.0 / (y + z));
  // Glorot & Bengio (AISTATS 2010)
  rng_uniform_gpu(y * z, -init_range, init_range, d_W);
  //CUDA_CHECK(hipMalloc((void**)&layer::d_weight_grad, y * z * sizeof(float_t)));
  float_malloc_device(y*z, layer::d_weight_grad);
  //CUDA_CHECK(hipMemset(layer::d_weight_grad, 0, y * z * sizeof(float_t)));
  init_const_gpu(y*z, 0.0, layer::d_weight_grad);
}

void graph_conv_layer::aggregate(size_t len, CSRGraph& g, const float_t* in, float_t* out) {
  #ifdef USE_CUSPARSE
  deepgalois::update_all_csrmm(len, g, in, out, norm_, norm_factor);
  #else
  deepgalois::update_all(len, g, in, out, norm_, norm_factor);
  #endif
}

void graph_conv_layer::d_aggregate(size_t len, CSRGraph& g, const float_t* in, float_t* out) {
#ifdef USE_CUSPARSE
  deepgalois::update_all_csrmm(len, g, in, out, norm_, norm_factor);
#else
  deepgalois::update_all(len, g, in, out, norm_, norm_factor);
#endif
}

void graph_conv_layer::combine(size_t dim_x, size_t dim_y, const float_t* self, const float_t* neighbors, float_t* out) {
}

// GPU forward: compute output features
// NOTE: in_data will be used in back-prop, so it can not be modified
void graph_conv_layer::forward_propagation(const float_t* in_data, float_t* out_data) {
  assert(z <= MAX_NUM_CLASSES); // currently only support feature length <= 128
  init_const_gpu(x*z, 0.0, out_temp);
  if (dropout_ && phase_ == deepgalois::net_phase::train)
    dropout_gpu(x * y, scale_, dropout_rate_, in_data, dropout_mask, in_temp);
  else copy_gpu(x*y, in_data, in_temp); 
  if (y > z) {
    sgemm_gpu(CblasNoTrans, CblasNoTrans, x, z, y, 1.0, in_temp, d_W, 0.0, out_temp);
    graph_conv_layer::aggregate(z, context->graph_gpu, out_temp, out_data);
  } else {
    graph_conv_layer::aggregate(y, context->graph_gpu, in_temp, in_temp1);
    sgemm_gpu(CblasNoTrans, CblasNoTrans, x, z, y, 1.0, in_temp1, d_W, 0.0, out_data);
  }
  if (act_) relu_gpu(x * z, out_data, out_data);
}

// GPU backward: compute input gradients (in_grad) and weight gradients (d_weight_grad)
void graph_conv_layer::back_propagation(const float_t* in_data,
                                        const float_t* out_data,
                                        float_t* out_grad, float_t* in_grad) {
  if (act_) d_relu_gpu(x * z, out_grad, out_data, out_grad);
  if (y > z) {
    graph_conv_layer::d_aggregate(z, context->graph_gpu, out_grad, out_temp);
    if (level_ != 0)
      sgemm_gpu(CblasNoTrans, CblasTrans, x, y, z, 1.0, out_temp, d_W, 0.0, in_grad);
    sgemm_gpu(CblasTrans, CblasNoTrans, y, z, x, 1.0, in_data, out_temp, 0.0, layer::d_weight_grad);
  } else {
    if (level_ != 0) {
      sgemm_gpu(CblasNoTrans, CblasTrans, x, y, z, 1.0, out_grad, d_W, 0.0, in_temp);
      graph_conv_layer::d_aggregate(y, context->graph_gpu, in_temp, in_grad);
    }
    sgemm_gpu(CblasTrans, CblasNoTrans, y, z, x, 1.0, in_data, out_grad, 0.0, layer::d_weight_grad);
  }
  if (level_ != 0 && dropout_)
    d_dropout_gpu(x * y, scale_, dropout_rate_, in_grad, dropout_mask, in_grad);
}

} // namespace

