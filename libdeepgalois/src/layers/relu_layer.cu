#include "hip/hip_runtime.h"
#include "deepgalois/layers/relu_layer.h"

namespace deepgalois {

// 𝑦[𝑙] = max(0, 𝑦[𝑙−1])
void relu_layer::forward_propagation(const float_t* in_data, float_t* out_data) {
  const size_t count = input_dims[0] * input_dims[1];
  relu_gpu(count, in_data, out_data);
}

// 𝜕𝐿 / 𝜕𝑦[𝑙−1] = 0, 𝑖𝑓 (𝑦[𝑙] < 0)
//              = 𝜕𝐿 / 𝜕𝑦𝑙, 𝑜𝑡ℎ𝑒𝑟𝑤𝑖𝑠𝑒
void relu_layer::back_propagation(const float_t* in_data, const float_t* out_data,
                                  float_t* out_grad, float_t* in_grad) {
  const size_t count = input_dims[0] * input_dims[1];
  d_relu_gpu(count, out_grad, in_data, in_grad);
}

} // namespace
