#include "node.h"
#include "cutils.h"

void edge::alloc_gpu() {
  CUDA_CHECK(
      hipMalloc((void**)&data_, num_samples_ * ft_dim_ * sizeof(float_t)));
  CUDA_CHECK(
      hipMalloc((void**)&grad_, num_samples_ * ft_dim_ * sizeof(float_t)));
}

void edge::merge_grads_gpu(float_t* dst) {
  CUDA_CHECK(hipMemcpy(&dst, grad_, ft_dim_ * sizeof(float_t),
                        hipMemcpyDeviceToHost));
}

void edge::clear_grads_gpu() {
  CUDA_CHECK(hipMemset(grad_, 0, ft_dim_ * num_samples_ * sizeof(float_t)));
}
