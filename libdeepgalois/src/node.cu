#include "deepgalois/layers/node.h"
#include "deepgalois/cutils.h"
#include "deepgalois/math_functions.hh"

namespace deepgalois {

void edge::alloc() {
  CUDA_CHECK(
      hipMalloc((void**)&data_, num_samples_ * ft_dim_ * sizeof(float_t)));
  CUDA_CHECK(
      hipMalloc((void**)&grad_, num_samples_ * ft_dim_ * sizeof(float_t)));
}

void edge::merge_grads(float_t* dst) {
  CUDA_CHECK(hipMemcpy(&dst, grad_, ft_dim_ * sizeof(float_t),
                        hipMemcpyDeviceToHost));
}

void edge::clear_grads() {
  // CUDA_CHECK(hipMemset(grad_, 0, num_samples_ * ft_dim_ * sizeof(float_t)));
  init_const_gpu(num_samples_ * ft_dim_, 0.0, grad_);
}

} // namespace deepgalois
