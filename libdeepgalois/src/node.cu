#include "deepgalois/layers/node.h"
#include "deepgalois/cutils.h"

void deepgalois::edge::alloc_gpu() {
  CUDA_CHECK(
      hipMalloc((void**)&data_, num_samples_ * ft_dim_ * sizeof(float_t)));
  CUDA_CHECK(
      hipMalloc((void**)&grad_, num_samples_ * ft_dim_ * sizeof(float_t)));
}

void deepgalois::edge::merge_grads_gpu(float_t* dst) {
  CUDA_CHECK(hipMemcpy(&dst, grad_, ft_dim_ * sizeof(float_t),
                        hipMemcpyDeviceToHost));
}

void deepgalois::edge::clear_grads_gpu() {
  CUDA_CHECK(hipMemset(grad_, 0, num_samples_ * ft_dim_ * sizeof(float_t)));
}
