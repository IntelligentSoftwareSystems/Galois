#include "deepgalois/lgraph.h"
#include "deepgalois/cutils.h"
#include <cassert>

namespace deepgalois {

void LearningGraph::dealloc() {
  assert(is_device);
  CUDA_CHECK(hipFree(colidx_));
  CUDA_CHECK(hipFree(rowptr_));
  CUDA_CHECK(hipFree(degrees_));
  if (edge_data_ != NULL) CUDA_CHECK(hipFree(edge_data_));
  if (vertex_data_ != NULL) CUDA_CHECK(hipFree(vertex_data_));
}

void LearningGraph::allocOnDevice(bool no_edge_data__) {
  if (colidx_ != NULL) return;  
  CUDA_CHECK(hipMalloc((void **) &colidx_, num_edges_ * sizeof(index_t)));
  CUDA_CHECK(hipMalloc((void **) &rowptr_, (num_vertices_+1) * sizeof(index_t)));
  CUDA_CHECK(hipMalloc((void **) &degrees_, num_vertices_ * sizeof(index_t)));
  //if (!no_edge_data__) CUDA_CHECK(hipMalloc((void **) &edge_data__, num_edges_ * sizeof(edge_data___t)));
  //CUDA_CHECK(hipMalloc((void **) &vertex_data__, num_vertices_ * sizeof(vdata_t)));
  is_device = true;
}

void LearningGraph::copy_to_gpu(LearningGraph &copygraph) {
  copygraph.init(num_vertices_, num_edges_);
  copygraph.allocOnDevice(edge_data_ == NULL);
  CUDA_CHECK(hipMemcpy(copygraph.colidx_, colidx_, num_edges_ * sizeof(index_t), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(copygraph.rowptr_, rowptr_, (num_vertices_+1) * sizeof(index_t), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(copygraph.degrees_, degrees_, num_vertices_ * sizeof(index_t), hipMemcpyHostToDevice));
  //if (edge_data__ != NULL) CUDA_CHECK(hipMemcpy(copygraph.edge_data__, edge_data__, num_edges_ * sizeof(edata_t), hipMemcpyHostToDevice));
  //CUDA_CHECK(hipMemcpy(copygraph.vertex_data__, vertex_data__, num_vertices_ * sizeof(vdata_t), hipMemcpyHostToDevice));
}

void LearningGraph::copy_to_cpu(LearningGraph &copygraph) {
  assert(is_device);
  assert(copygraph.size() == num_vertices_);
  assert(copygraph.sizeEdges() == num_edges_);
  CUDA_CHECK(hipMemcpy(copygraph.edge_dst_ptr(), colidx_, num_edges_ * sizeof(index_t), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(copygraph.row_start_ptr(), rowptr_, (num_vertices_+1) * sizeof(index_t), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(copygraph.degrees_ptr(), degrees_, num_vertices_ * sizeof(index_t), hipMemcpyDeviceToHost));
  //if (edge_data__ != NULL) CUDA_CHECK(hipMemcpy(copygraph.edge_data__ptr(), edge_data__, num_edges_ * sizeof(edata_t), hipMemcpyDeviceToHost));
  //CUDA_CHECK(hipMemcpy(copygraph.vertex_data__ptr(), vertex_data__, num_vertices_ * sizeof(vdata_t), hipMemcpyDeviceToHost));
}

}
