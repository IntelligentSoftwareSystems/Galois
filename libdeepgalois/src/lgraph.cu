
void LearningGraph::dealloc() {
  assert(is_device);
  CUDA_CHECK(hipFree(colidx_));
  CUDA_CHECK(hipFree(rowptr_));
  CUDA_CHECK(hipFree(degrees_));
  if (edge_data != NULL) CUDA_CHECK(hipFree(edge_data));
  if (vertex_data != NULL) CUDA_CHECK(hipFree(vertex_data));
}

void LearningGraph::allocOnDevice(bool no_edge_data_) {
  if (colidx_ != NULL) return true;  
  CUDA_CHECK(hipMalloc((void **) &colidx_, num_edges_ * sizeof(index_type)));
  CUDA_CHECK(hipMalloc((void **) &rowptr_, (num_vertices_+1) * sizeof(index_type)));
  CUDA_CHECK(hipMalloc((void **) &degrees_, num_vertices_ * sizeof(index_type)));
  //if (!no_edge_data_) CUDA_CHECK(hipMalloc((void **) &edge_data_, num_edges_ * sizeof(edge_data__t)));
  //CUDA_CHECK(hipMalloc((void **) &vertex_data_, num_vertices_ * sizeof(vdata_t)));
  is_device = true;
}

void LearningGraph::copy_to_gpu(LearningGraph &copygraph) {
  copygraph.init(num_vertices_, num_edges_);
  copygraph.allocOnDevice(edge_data_ == NULL);
  CUDA_CHECK(hipMemcpy(copygraph.colidx_, colidx_, num_edges_ * sizeof(index_type), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(copygraph.rowptr_, rowptr_, (num_vertices_+1) * sizeof(index_type), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(copygraph.degrees_, degrees_, num_vertices_ * sizeof(index_type), hipMemcpyHostToDevice));
  //if (edge_data_ != NULL) CUDA_CHECK(hipMemcpy(copygraph.edge_data_, edge_data_, num_edges_ * sizeof(edata_t), hipMemcpyHostToDevice));
  //CUDA_CHECK(hipMemcpy(copygraph.vertex_data_, vertex_data_, num_vertices_ * sizeof(vdata_t), hipMemcpyHostToDevice));
}

void LearningGraph::copy_to_cpu(LearningGraph &copygraph) {
  assert(is_device);
  assert(copygraph.size() = num_vertices_);
  assert(copygraph.sizeEdges() = num_edges_);
  CUDA_CHECK(hipMemcpy(copygraph.edge_dst_ptr(), colidx_, num_edges_ * sizeof(index_type), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(copygraph.row_start_ptr(), rowptr_, (num_vertices_+1) * sizeof(index_type), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(copygraph.degrees_ptr(), degrees_, num_vertices_ * sizeof(index_type), hipMemcpyDeviceToHost));
  //if (edge_data_ != NULL) CUDA_CHECK(hipMemcpy(copygraph.edge_data_ptr(), edge_data_, num_edges_ * sizeof(edata_t), hipMemcpyDeviceToHost));
  //CUDA_CHECK(hipMemcpy(copygraph.vertex_data_ptr(), vertex_data_, num_vertices_ * sizeof(vdata_t), hipMemcpyDeviceToHost));
}

