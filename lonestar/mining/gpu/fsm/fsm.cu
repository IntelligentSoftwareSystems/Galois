#include "hip/hip_runtime.h"
// Copyright (c) 2019, Xuhao Chen
#include "fsm.h"
#include "pangolin/timer.h"
#include "pangolin/cutils.h"
#define USE_PID
#define USE_DOMAIN
#define EDGE_INDUCED
#define ENABLE_LABEL
#include <hipcub/hipcub.hpp>
#include "pangolin/miner.cuh"
#include "pangolin/bitsets.h"
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#define MAX_NUM_PATTERNS 21251

struct OrderedEdge {
  IndexT src;
  IndexT dst;
};

inline __device__ int get_init_pattern_id(node_data_type src_label,
                                          node_data_type dst_label,
                                          int nlabels) {
  return (int)src_label * nlabels + (int)dst_label;
}

inline __device__ unsigned get_pattern_id(node_data_type label0,
                                          node_data_type label1,
                                          node_data_type label2, int nlabels) {
  return nlabels * (nlabels * label0 + label1) + label2;
}

inline __device__ bool is_quick_automorphism(unsigned size, IndexT* vids,
                                             history_type his2,
                                             history_type his, IndexT src,
                                             IndexT dst) {
  if (dst <= vids[0])
    return true;
  if (dst == vids[1])
    return true;
  if (his == 0 && dst < vids[1])
    return true;
  if (size == 2) {
  } else if (size == 3) {
    if (his == 0 && his2 == 0 && dst <= vids[2])
      return true;
    if (his == 0 && his2 == 1 && dst == vids[2])
      return true;
    if (his == 1 && his2 == 1 && dst <= vids[2])
      return true;
  } else {
  }
  return false;
}

inline __device__ void swap(IndexT first, IndexT second) {
  if (first > second) {
    IndexT tmp = first;
    first      = second;
    second     = tmp;
  }
}

inline __device__ int compare(OrderedEdge oneEdge, OrderedEdge otherEdge) {
  swap(oneEdge.src, oneEdge.dst);
  swap(otherEdge.src, otherEdge.dst);
  if (oneEdge.src == otherEdge.src)
    return oneEdge.dst - otherEdge.dst;
  else
    return oneEdge.src - otherEdge.src;
}

inline __device__ bool is_edge_automorphism(unsigned size, IndexT* vids,
                                            history_type* hiss,
                                            history_type his, IndexT src,
                                            IndexT dst) {
  if (size < 3)
    return is_quick_automorphism(size, vids, hiss[2], his, src, dst);
  if (dst <= vids[0])
    return true;
  if (his == 0 && dst <= vids[1])
    return true;
  if (dst == vids[hiss[his]])
    return true;
  OrderedEdge added_edge;
  added_edge.src = src;
  added_edge.dst = dst;
  for (unsigned index = his + 1; index < size; ++index) {
    OrderedEdge edge;
    edge.src = vids[hiss[index]];
    edge.dst = vids[index];
    int cmp  = compare(added_edge, edge);
    if (cmp <= 0)
      return true;
  }
  return false;
}

__global__ void extend_alloc(unsigned m, unsigned level, CSRGraph graph,
                             EmbeddingList emb_list, IndexT* num_new_emb) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vid[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  __shared__ history_type his[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  if (pos < m) {
    emb_list.get_edge_embedding(level, pos, vid[tid], his[tid]);
    num_new_emb[pos] = 0;
    // if (pos == 1) printf("src=%d, dst=%d\n", vid[tid][0], vid[tid][1]);
    for (unsigned i = 0; i < level + 1; ++i) {
      IndexT src       = vid[tid][i];
      IndexT row_begin = graph.edge_begin(src);
      IndexT row_end   = graph.edge_end(src);
      for (IndexT e = row_begin; e < row_end; e++) {
        IndexT dst = graph.getEdgeDst(e);
        if (!is_edge_automorphism(level + 1, vid[tid], his[tid], i, src, dst))
          num_new_emb[pos]++;
      }
    }
  }
}

__global__ void extend_insert(unsigned m, unsigned level, CSRGraph graph,
                              EmbeddingList emb_list, IndexT* indices) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vids[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  __shared__ history_type his[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  if (pos < m) {
    emb_list.get_edge_embedding(level, pos, vids[tid], his[tid]);
    IndexT start = indices[pos];
    for (unsigned i = 0; i < level + 1; ++i) {
      IndexT src       = vids[tid][i];
      IndexT row_begin = graph.edge_begin(src);
      IndexT row_end   = graph.edge_end(src);
      for (IndexT e = row_begin; e < row_end; e++) {
        IndexT dst = graph.getEdgeDst(e);
        if (!is_edge_automorphism(level + 1, vids[tid], his[tid], i, src,
                                  dst)) {
          emb_list.set_idx(level + 1, start, pos);
          emb_list.set_his(level + 1, start, i);
          emb_list.set_vid(level + 1, start++, dst);
        }
      }
    }
  }
}

__global__ void init_aggregate(unsigned m, unsigned num_emb, CSRGraph graph,
                               EmbeddingList emb_list, unsigned* pids,
                               int nlabels, unsigned threshold,
                               Bitsets small_sets, Bitsets large_sets) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if (pos < num_emb) {
    IndexT src               = emb_list.get_idx(1, pos);
    IndexT dst               = emb_list.get_vid(1, pos);
    node_data_type src_label = graph.getData(src);
    node_data_type dst_label = graph.getData(dst);
    int pid                  = 0;
    if (src_label <= dst_label)
      pid = get_init_pattern_id(src_label, dst_label, nlabels);
    else
      pid = get_init_pattern_id(dst_label, src_label, nlabels);
    pids[pos] = pid;
    if (src_label < dst_label) {
      small_sets.set(pid, src);
      large_sets.set(pid, dst);
    } else if (src_label > dst_label) {
      small_sets.set(pid, dst);
      large_sets.set(pid, src);
    } else {
      small_sets.set(pid, src);
      small_sets.set(pid, dst);
      large_sets.set(pid, src);
      large_sets.set(pid, dst);
    }
  }
}

__global__ void count_ones(int id, Bitsets sets, int* count) {
  typedef hipcub::BlockReduce<int, BLOCK_SIZE> BlockReduce;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int num = 0;
  if (pos < sets.vec_size())
    num = sets.count_num_ones(id, pos);
  int block_total = BlockReduce(temp_storage).Sum(num);
  if (threadIdx.x == 0)
    atomicAdd(count, block_total);
}

int init_support_count(unsigned m, int npatterns, unsigned threshold,
                       Bitsets small_sets, Bitsets large_sets,
                       bool* init_support_map) {
  int num_freq_patterns = 0;
  for (int i = 0; i < npatterns; i++) {
    int a, b, *d_count;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_count, sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m - 1) / 256 + 1, 256>>>(i, small_sets, d_count);
    CudaTest("solving count_ones `failed");
    CUDA_SAFE_CALL(
        hipMemcpy(&a, d_count, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m - 1) / 256 + 1, 256>>>(i, large_sets, d_count);
    CUDA_SAFE_CALL(
        hipMemcpy(&b, d_count, sizeof(int), hipMemcpyDeviceToHost));
    unsigned support = a < b ? a : b;
    if (support >= threshold) {
      init_support_map[i] = 1;
      num_freq_patterns++;
    } else
      init_support_map[i] = 0;
  }
  return num_freq_patterns;
}

int support_count(unsigned m, unsigned npatterns, unsigned threshold,
                  Bitsets small_sets, Bitsets middle_sets, Bitsets large_sets,
                  bool* support_map) {
  int num_freq_patterns = 0;
  for (int i = 0; i < npatterns; i++) {
    int a, b, c, *d_count;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_count, sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m - 1) / 256 + 1, 256>>>(i, small_sets, d_count);
    CUDA_SAFE_CALL(
        hipMemcpy(&a, d_count, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m - 1) / 256 + 1, 256>>>(i, large_sets, d_count);
    CUDA_SAFE_CALL(
        hipMemcpy(&b, d_count, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m - 1) / 256 + 1, 256>>>(i, middle_sets, d_count);
    CUDA_SAFE_CALL(
        hipMemcpy(&c, d_count, sizeof(int), hipMemcpyDeviceToHost));
    unsigned small   = a < b ? a : b;
    unsigned support = small < c ? small : c;
    if (support >= threshold) {
      support_map[i] = 1;
      num_freq_patterns++;
    } else
      support_map[i] = 0;
  }
  return num_freq_patterns;
}

__global__ void init_filter_check(unsigned m, unsigned* pids,
                                  bool* init_support_map,
                                  IndexT* is_frequent_emb) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if (pos < m) {
    unsigned pid     = pids[pos];
    bool is_frequent = init_support_map[pid];
    if (is_frequent)
      is_frequent_emb[pos] = 1;
  }
}

__global__ void copy_vids(unsigned m, EmbeddingList emb_list, IndexT* vid_list0,
                          IndexT* vid_list1) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if (pos < m) {
    vid_list0[pos] = emb_list.get_idx(1, pos);
    vid_list1[pos] = emb_list.get_vid(1, pos);
  }
}

__global__ void init_filter(unsigned m, EmbeddingList emb_list,
                            IndexT* vid_list0, IndexT* vid_list1,
                            IndexT* indices, IndexT* is_frequent_emb) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if (pos < m) {
    if (is_frequent_emb[pos]) {
      IndexT src     = vid_list0[pos];
      IndexT dst     = vid_list1[pos];
      unsigned start = indices[pos];
      emb_list.set_vid(1, start, dst);
      emb_list.set_idx(1, start, src);
    }
  }
}

__global__ void aggregate_check(unsigned num_emb, unsigned level,
                                CSRGraph graph, EmbeddingList emb_list,
                                unsigned* pids, int nlabels, unsigned threshold,
                                unsigned* ne) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vids[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  __shared__ history_type his[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  if (pos < num_emb) {
    emb_list.get_edge_embedding(level, pos, vids[tid], his[tid]);
    unsigned n = level + 1;
    assert(n < 4);
    IndexT first      = vids[tid][0];
    IndexT second     = vids[tid][1];
    IndexT third      = vids[tid][2];
    node_data_type l0 = graph.getData(first);
    node_data_type l1 = graph.getData(second);
    node_data_type l2 = graph.getData(third);
    history_type h2   = his[tid][2];
    unsigned pid      = 0;
    if (n == 3) {
      if (h2 == 0) {
        if (l1 < l2) {
          pid = get_pattern_id(l0, l2, l1, nlabels);
        } else {
          pid = get_pattern_id(l0, l1, l2, nlabels);
        }
      } else {
        assert(h2 == 1);
        if (l0 < l2) {
          pid = get_pattern_id(l1, l2, l0, nlabels);
        } else {
          pid = get_pattern_id(l1, l0, l2, nlabels);
        }
      }
    } else {
    }
    pids[pos] = pid;
    atomicAdd(&ne[pid], 1);
  }
}

__global__ void find_candidate_patterns(unsigned num_patterns, unsigned* ne,
                                        unsigned minsup, unsigned* id_map,
                                        unsigned* num_new_patterns) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if (pos < num_patterns) {
    if (ne[pos] >= minsup) {
      unsigned new_id = atomicAdd(num_new_patterns, 1);
      id_map[pos]     = new_id;
    }
  }
}

__global__ void aggregate(unsigned m, unsigned num_emb, unsigned level,
                          CSRGraph graph, EmbeddingList emb_list,
                          unsigned* pids, unsigned* ne, unsigned* id_map,
                          int nlabels, unsigned threshold, Bitsets small_sets,
                          Bitsets middle_sets, Bitsets large_sets) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vids[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  __shared__ history_type his[BLOCK_SIZE][PANGOLIN_MAX_SIZE];
  if (pos < num_emb) {
    emb_list.get_edge_embedding(level, pos, vids[tid], his[tid]);
    assert(level == 2);
    IndexT first      = vids[tid][0];
    IndexT second     = vids[tid][1];
    IndexT third      = vids[tid][2];
    node_data_type l0 = graph.getData(first);
    node_data_type l1 = graph.getData(second);
    node_data_type l2 = graph.getData(third);
    history_type h2   = his[tid][2];
    IndexT small, middle, large;
    unsigned pid = pids[pos];
    if (ne[pid] >= threshold) {
      pid = id_map[pid];
      if (h2 == 0) {
        middle = first;
        if (l1 < l2) {
          small = second;
          large = third;
        } else {
          small = third;
          large = second;
        }
        small_sets.set(pid, small);
        middle_sets.set(pid, middle);
        large_sets.set(pid, large);
        if (l1 == l2) {
          small_sets.set(pid, large);
          large_sets.set(pid, small);
        }
      } else {
        assert(h2 == 1);
        middle = second;
        if (l0 < l2) {
          small = first;
          large = third;
        } else {
          small = third;
          large = first;
        }
        small_sets.set(pid, small);
        middle_sets.set(pid, middle);
        large_sets.set(pid, large);
        if (l0 == l2) {
          small_sets.set(pid, large);
          large_sets.set(pid, small);
        }
      }
    }
  }
}

void parallel_prefix_sum(int n, IndexT* in, IndexT* out) {
  IndexT total = 0;
  for (size_t i = 0; i < n; i++) {
    out[i] = total;
    total += in[i];
  }
  out[n] = total;
}

void fsm_gpu_solver(std::string fname, unsigned k, unsigned minsup,
                    AccType& total_num) {
  CSRGraph graph_cpu, graph_gpu;
  int nlabels = graph_cpu.read(fname); // read graph into CPU memoryA
  int m       = graph_cpu.get_nnodes();
  int nnz     = graph_cpu.get_nedges();
  graph_cpu.copy_to_gpu(graph_gpu); // copy graph to GPU memory
  EmbeddingList emb_list;
  emb_list.init(nnz, k + 1, false);
  emb_list.init_cpu(&graph_cpu);

  int nthreads          = BLOCK_SIZE;
  int nblocks           = DIVIDE_INTO(nnz, nthreads);
  int num_init_patterns = (nlabels + 1) * (nlabels + 1);
  std::cout << "Number of init patterns: " << num_init_patterns << std::endl;
  unsigned num_emb = emb_list.size();
  std::cout << "number of single-edge embeddings: " << num_emb << "\n";
  unsigned* pids;
  CUDA_SAFE_CALL(hipMalloc((void**)&pids, sizeof(unsigned) * num_emb));
  bool* h_init_support_map = (bool*)malloc(sizeof(bool) * num_init_patterns);
  bool* d_init_support_map;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_init_support_map,
                            sizeof(bool) * num_init_patterns));
  IndexT* is_frequent_emb;
  CUDA_SAFE_CALL(
      hipMalloc((void**)&is_frequent_emb, sizeof(IndexT) * (num_emb + 1)));
  CUDA_SAFE_CALL(
      hipMemset(is_frequent_emb, 0, sizeof(IndexT) * (num_emb + 1)));
  IndexT *vid_list0, *vid_list1;
  CUDA_SAFE_CALL(hipMalloc((void**)&vid_list0, sizeof(IndexT) * num_emb));
  CUDA_SAFE_CALL(hipMalloc((void**)&vid_list1, sizeof(IndexT) * num_emb));
  Bitsets small_sets, large_sets, middle_sets;
  small_sets.alloc(MAX_NUM_PATTERNS, m);
  large_sets.alloc(MAX_NUM_PATTERNS, m);
  middle_sets.alloc(MAX_NUM_PATTERNS, m);
  small_sets.set_size(num_init_patterns, m);
  large_sets.set_size(num_init_patterns, m);
  middle_sets.set_size(num_init_patterns, m);

  IndexT *num_new_emb, *indices;
  CUDA_SAFE_CALL(hipMalloc((void**)&indices, sizeof(IndexT) * (num_emb + 1)));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  nblocks = (num_emb - 1) / nthreads + 1;
  unsigned* d_num_new_patterns;
  unsigned h_num_new_patterns = 0;
  CUDA_SAFE_CALL(hipMalloc((void**)&d_num_new_patterns, sizeof(unsigned)));
  printf("Launching CUDA TC solver (%d CTAs, %d threads/CTA) ...\n", nblocks,
         nthreads);

  Timer t;
  t.Start();
  unsigned level = 1;
  init_aggregate<<<nblocks, nthreads>>>(m, num_emb, graph_gpu, emb_list, pids,
                                        nlabels, minsup, small_sets,
                                        large_sets);
  CudaTest("solving init_aggregate `failed");
  std::cout << "Init_aggregate Done\n";
  int num_freq_patterns = init_support_count(
      m, num_init_patterns, minsup, small_sets, large_sets, h_init_support_map);
  total_num += num_freq_patterns;
  if (num_freq_patterns == 0) {
    std::cout << "No frequent pattern found\n\n";
    return;
  }
  std::cout << "Number of frequent single-edge patterns: " << num_freq_patterns
            << "\n";
  CUDA_SAFE_CALL(hipMemcpy(d_init_support_map, h_init_support_map,
                            sizeof(bool) * num_init_patterns,
                            hipMemcpyHostToDevice));
  init_filter_check<<<nblocks, nthreads>>>(num_emb, pids, d_init_support_map,
                                           is_frequent_emb);
  CudaTest("solving init_filter_check `failed");
  thrust::exclusive_scan(thrust::device, is_frequent_emb,
                         is_frequent_emb + num_emb + 1, indices);
  IndexT new_size;
  CUDA_SAFE_CALL(hipMemcpy(&new_size, &indices[num_emb], sizeof(IndexT),
                            hipMemcpyDeviceToHost));
  std::cout << "number of embeddings after pruning: " << new_size << "\n";
  copy_vids<<<nblocks, nthreads>>>(num_emb, emb_list, vid_list0, vid_list1);
  CudaTest("solving copy_vids `failed");
  init_filter<<<nblocks, nthreads>>>(num_emb, emb_list, vid_list0, vid_list1,
                                     indices, is_frequent_emb);
  CudaTest("solving init_filter `failed");
  CUDA_SAFE_CALL(hipFree(indices));
  CUDA_SAFE_CALL(hipFree(is_frequent_emb));
  CUDA_SAFE_CALL(hipFree(pids));
  // small_sets.clean();
  // large_sets.clean();
  small_sets.clear();
  large_sets.clear();
  CUDA_SAFE_CALL(hipFree(vid_list0));
  CUDA_SAFE_CALL(hipFree(vid_list1));
  CUDA_SAFE_CALL(hipFree(d_init_support_map));
  emb_list.remove_tail(new_size);

  while (1) {
    num_emb = emb_list.size();
    std::cout << "number of embeddings in level " << level << ": " << num_emb
              << "\n";
    CUDA_SAFE_CALL(
        hipMalloc((void**)&num_new_emb, sizeof(IndexT) * (num_emb + 1)));
    CUDA_SAFE_CALL(
        hipMalloc((void**)&indices, sizeof(IndexT) * (num_emb + 1)));
    std::cout << "Done allocating memory for embeddings in level " << level
              << "\n";
    nblocks = (num_emb - 1) / nthreads + 1;
    extend_alloc<<<nblocks, nthreads>>>(num_emb, level, graph_gpu, emb_list,
                                        num_new_emb);
    CudaTest("solving extend_alloc failed");
    thrust::exclusive_scan(thrust::device, num_new_emb,
                           num_new_emb + num_emb + 1, indices);
    CudaTest("Scan failed");
    CUDA_SAFE_CALL(hipMemcpy(&new_size, &indices[num_emb], sizeof(IndexT),
                              hipMemcpyDeviceToHost));
    std::cout << "number of new embeddings: " << new_size << "\n";
    emb_list.add_level(new_size);
    extend_insert<<<nblocks, nthreads>>>(num_emb, level, graph_gpu, emb_list,
                                         indices);
    CudaTest("solving extend_insert failed");
    std::cout << "Extend_insert Done\n";
    num_emb = emb_list.size();
    CUDA_SAFE_CALL(hipFree(num_new_emb));
    CUDA_SAFE_CALL(hipFree(indices));
    level++;

    int num_patterns = nlabels * num_init_patterns;
    nblocks          = (num_emb - 1) / nthreads + 1;
    std::cout << "Number of patterns in level " << level << ": " << num_patterns
              << std::endl;
    std::cout << "number of embeddings in level " << level << ": " << num_emb
              << "\n";
    unsigned *ne, *id_map;
    CUDA_SAFE_CALL(hipMalloc((void**)&ne, sizeof(unsigned) * num_patterns));
    CUDA_SAFE_CALL(
        hipMalloc((void**)&id_map, sizeof(unsigned) * num_patterns));
    CUDA_SAFE_CALL(hipMemset(ne, 0, sizeof(unsigned) * num_patterns));
    CUDA_SAFE_CALL(hipMalloc((void**)&pids, sizeof(unsigned) * num_emb));
    std::cout << "Done allocating memory for aggregation in level " << level
              << "\n";
    aggregate_check<<<nblocks, nthreads>>>(num_emb, level, graph_gpu, emb_list,
                                           pids, nlabels, minsup, ne);
    CudaTest("solving aggregate_check failed");
    CUDA_SAFE_CALL(hipMemset(d_num_new_patterns, 0, sizeof(unsigned)));
    find_candidate_patterns<<<(num_patterns - 1) / nthreads + 1, nthreads>>>(
        num_patterns, ne, minsup, id_map, d_num_new_patterns);
    CudaTest("solving find_candidate_patterns failed");
    CUDA_SAFE_CALL(hipMemcpy(&h_num_new_patterns, d_num_new_patterns,
                              sizeof(unsigned), hipMemcpyDeviceToHost));
    std::cout << "Number of candidate patterns in level " << level << ": "
              << h_num_new_patterns << std::endl;

    // small_sets.alloc(h_num_new_patterns, m);
    // large_sets.alloc(h_num_new_patterns, m);
    // middle_sets.alloc(h_num_new_patterns, m);
    small_sets.set_size(h_num_new_patterns, m);
    large_sets.set_size(h_num_new_patterns, m);
    middle_sets.set_size(h_num_new_patterns, m);
    std::cout << "Done allocating sets\n";
    aggregate<<<nblocks, nthreads>>>(m, num_emb, level, graph_gpu, emb_list,
                                     pids, ne, id_map, nlabels, minsup,
                                     small_sets, middle_sets, large_sets);
    CudaTest("solving aggregate failed");
    bool* h_support_map = (bool*)malloc(sizeof(bool) * h_num_new_patterns);
    num_freq_patterns = support_count(m, h_num_new_patterns, minsup, small_sets,
                                      middle_sets, large_sets, h_support_map);
    CudaTest("solving support_count failed");
    CUDA_SAFE_CALL(hipFree(ne));
    CUDA_SAFE_CALL(hipFree(id_map));
    std::cout << "num_frequent_patterns: " << num_freq_patterns << "\n";
    total_num += num_freq_patterns;
    if (num_freq_patterns == 0)
      break;
    if (level == k)
      break;
    // filter<<<nblocks, nthreads>>>(level, emb_list);
  }
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  printf("\truntime = %f ms.\n", t.Millisecs());
}
