#include "galois/CUDAUtil.h"
#include "galois/graphs/GNNGraph.cuh"

void galois::graphs::GNNGraphGPUAllocations::SetFeatures(
    const std::vector<GNNFeature>& features) {
  CUDA_CHECK(hipMalloc((void**)(&feature_vector_),
                        features.size() * sizeof(GNNFeature)));
  CUDA_CHECK(hipMemcpy(feature_vector_, features.data(),
                        features.size() * sizeof(GNNFeature),
                        hipMemcpyHostToDevice));
}
